#include "hip/hip_runtime.h"
#include "HinesMatrix.hpp"
#include "PlatformFunctions.hpp"
#include "HinesStruct.hpp"
#include "Connections.hpp"
#include "SpikeStatistics.hpp"
#include "GpuSimulationControl.hpp"

#include "SynapticData.hpp"
#include "KernelInfo.hpp"
#include "ThreadInfo.hpp"
#include "SharedNeuronGpuData.hpp"


#include <cassert>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <pthread.h>

#ifdef MPI_GPU_NN
#include <mpi.h>
#endif

#include <hip/hip_runtime.h> // Necessary to allow better eclipse integration
#include <hip/hip_runtime_api.h> // Necessary to allow better eclipse integration
#include <> // Necessary to allow better eclipse integration
#include <hip/device_functions.h> // Necessary to allow better eclipse integration

extern __global__ void solveMatrixG(HinesStruct *hList, int nSteps, int nNeurons, ftype *vmListGlobal);
extern ConnGpu* createGpuConnections( ConnectionInfo *connInfoList, int destType, int *nNeurons, int nGroups );
extern int **countReceivedSpikesCpu(ConnGpu *connGpuList, int nNeurons, int nGroups, ucomp **nGeneratedSpikes);

extern __global__ void performCommunicationsG(int nNeurons, ConnGpu *connGpuListDev,
		ucomp **nGeneratedSpikesDev, ftype **genSpikeTimeListDev, HinesStruct *hList,
		ftype *randomSpikeTimesDev, int *randomSpikeDestDev, int nRandom);

void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

//===================================================================================================

GpuSimulationControl::GpuSimulationControl(ThreadInfo *tInfo) {

	this->tInfo = tInfo;
	this->sharedData = tInfo->sharedData;
	this->kernelInfo = tInfo->sharedData->kernelInfo;
}

void GpuSimulationControl::prepareSynapses() {

	ftype spikeMem = 0;

	int *nNeurons = tInfo->nNeurons;
	SynapticData* synData = sharedData->synData;

	/**
	 * Prepare the synaptic channels and spike generation
	 */
	int totalTypes = synData->totalTypes;

	pthread_mutex_lock (sharedData->mutex);
	if (synData->activationListGlobal == 0) {

		synData->activationListGlobal    = (ftype **) malloc (sizeof(ftype *) * totalTypes); //*
		synData->activationListPosGlobal = (ucomp **) malloc (sizeof(ucomp *) * totalTypes); //*
		synData->activationListDevice    = (ftype **) malloc (sizeof(ftype *) * totalTypes); //*
		synData->activationListPosDevice = (ucomp **) malloc (sizeof(ucomp *) * totalTypes); //*
		if (benchConf.checkCommMode(NN_TEST) == NN_TEST)
			synData->activationListTest    = (ftype **) malloc (sizeof(ftype *) * totalTypes); //*


		synData->vmListHost    = (ftype **) malloc (sizeof(ftype *) * totalTypes);
		synData->vmListDevice  = (ftype **) malloc (sizeof(ftype *) * totalTypes);

		/*
		 * [MPI] Used in the CPU and GPU version to distribute the spike list among the processes
		 */
		synData->genSpikeTimeListHost     = (ftype **) malloc (sizeof(ftype *)  * totalTypes);
		synData->nGeneratedSpikesHost     = (ucomp **) malloc (sizeof(ucomp *)  * totalTypes);

		synData->genSpikeTimeListDevice   = (ftype **) malloc (sizeof(ftype *)  * totalTypes);
		synData->nGeneratedSpikesDevice   = (ucomp **) malloc (sizeof(ucomp *)  * totalTypes);

		if (benchConf.checkProcMode(NN_GPU) || benchConf.checkCommMode(NN_GPU) ) {
			synData->genSpikeTimeListGpusDev  = (ftype ***) malloc (sizeof(ftype **) * sharedData->nThreadsCpu);
			synData->genSpikeTimeListGpusHost = (ftype ***) malloc (sizeof(ftype **) * sharedData->nThreadsCpu);
			synData->nGeneratedSpikesGpusDev  = (ucomp ***) malloc (sizeof(ucomp **) * sharedData->nThreadsCpu);
			synData->nGeneratedSpikesGpusHost = (ucomp ***) malloc (sizeof(ucomp **) * sharedData->nThreadsCpu);
		}
	}
	pthread_mutex_unlock (sharedData->mutex);


	/**
	 * Prepare the delivered spike related lists
	 * - spikeListPos and spikeListSize
	 */
	for (int type = tInfo->startTypeThread; type < tInfo->endTypeThread; type++) {

		SynapticChannels *syn0 = sharedData->matrixList[type][0].synapticChannels;
		int globalActListSize = syn0->activationListSize * syn0->synapseListSize * nNeurons[type];
		synData->activationListGlobal[type] = (ftype *) malloc (sizeof(ftype) * globalActListSize); //*
		if (benchConf.checkCommMode(NN_TEST) == NN_TEST)
			synData->activationListTest[type] = (ftype *) malloc (sizeof(ftype) * globalActListSize);
		hipMalloc ((void **) &(synData->activationListDevice[type]), sizeof(ftype) * globalActListSize); //*
		for (int i=0; i < globalActListSize; i++)
			synData->activationListGlobal[type][i] = 0;
		hipMemcpy (synData->activationListDevice[type], synData->activationListGlobal[type],
				sizeof(ftype) * globalActListSize, hipMemcpyHostToDevice); //*


		synData->activationListPosGlobal[type] = (ucomp *) malloc (sizeof(ucomp) * syn0->synapseListSize * nNeurons[type]); //*
		hipMalloc ((void **) &(synData->activationListPosDevice[type]), sizeof(ucomp) * syn0->synapseListSize * nNeurons[type]); //*
		for (int i=0; i<syn0->synapseListSize * nNeurons[type]; i++)
			synData->activationListPosGlobal[type][i] = 0;
		hipMemcpy (synData->activationListPosDevice[type], synData->activationListPosGlobal[type],
				sizeof(ucomp) * syn0->synapseListSize * nNeurons[type], hipMemcpyHostToDevice); //*


		synData->vmListHost[type] = (ftype *) malloc(sizeof(ftype) * nNeurons[type]);
		hipMalloc ((void **) &(synData->vmListDevice[type]), sizeof(ftype)  * nNeurons[type]);

		spikeMem += sizeof(ftype) * (globalActListSize + nNeurons[type]) + sizeof(ucomp) * syn0->synapseListSize * nNeurons[type];
	}

	/**
	 * Prepare the lists containing the generated spikes during each kernel call
	 */
	for (int type = tInfo->startTypeThread; type < tInfo->endTypeThread; type++) {
		int spikeTimeListSize = GENSPIKETIMELIST_SIZE;

		synData->genSpikeTimeListHost[type] = (ftype *) malloc(sizeof(ftype) * spikeTimeListSize * nNeurons[type]);
		hipMalloc ((void **) &(synData->genSpikeTimeListDevice[type]), sizeof(ftype) * spikeTimeListSize * nNeurons[type]);

		synData->nGeneratedSpikesHost[type] = (ucomp *) malloc(sizeof(ucomp) * nNeurons[type]);
		hipMalloc ((void **) &(synData->nGeneratedSpikesDevice[type]), sizeof(ucomp) * nNeurons[type]);

		int synapseListSize = sharedData->matrixList[type][0].synapticChannels->synapseListSize;

		for (int neuron = 0; neuron < nNeurons[type]; neuron++ ) {
			HinesStruct & h = sharedData->hList[type][neuron];
			h.spikeTimes  = synData->genSpikeTimeListDevice[type] + spikeTimeListSize * neuron;
			h.nGeneratedSpikes = synData->nGeneratedSpikesDevice[type];// + neuron;

			h.activationList = synData->activationListDevice[type]; // global list
			h.activationListPos = synData->activationListPosDevice[type] + synapseListSize * neuron;
		}

		spikeMem += sizeof(ftype) * spikeTimeListSize * nNeurons[type] + sizeof(ucomp) * nNeurons[type];
	}

	printf("Memory for Synapses: %10.3f MB.\n", spikeMem/(1024.*1024.));

}

void GpuSimulationControl::testGpuCommunication(int type) {

	SynapticData *synData = sharedData->synData;

	int globalActListSize = sharedData->hList[type][0].synapseListSize * sharedData->hList[type][0].activationListSize * tInfo->nNeurons[type];
	hipMemcpy(synData->activationListTest[type], sharedData->hList[type][0].activationList,
			sizeof(ftype) * globalActListSize, hipMemcpyDeviceToHost);

	int neuronListSize = sharedData->hList[type][0].synapseListSize * sharedData->hList[type][0].activationListSize;

	for (int n=0; n<tInfo->nNeurons[type]; n++)
		for (int posN=0; posN<neuronListSize; posN++) {

			int nSyn        = sharedData->hList[type][0].synapseListSize;
			int actListSize = sharedData->hList[type][0].activationListSize;
			int synapse = posN / actListSize;
			int posSyn  = posN % actListSize;
			int posRel  = (synData->activationListPosGlobal[type][n*nSyn + synapse] + posSyn ) % actListSize;

			int s = posN * tInfo->nNeurons[type] + n;
			if ( fabs(synData->activationListTest[type][s] - synData->activationListGlobal[type][s]) > 0.001) {

				printf("type=%d neuron=%3d syn=%d, pos=%3d, posRel=%3d c=%5.2f|g=%5.2f\n", type, n, synapse, posSyn, posRel,
						synData->activationListGlobal[type][s], synData->activationListTest[type][s]);
				//assert (false);
			}
		}
}

void GpuSimulationControl::copyActivationListToGpu(int type) {

	int globalActListSize = sharedData->hList[type][0].synapseListSize * sharedData->hList[type][0].activationListSize * tInfo->nNeurons[type];
	hipMemcpy(sharedData->hList[type][0].activationList, sharedData->synData->activationListGlobal[type],
			sizeof(ftype) * globalActListSize, hipMemcpyHostToDevice);

}

void GpuSimulationControl::copyActivationListFromGpu() {

	SynapticData *synData = sharedData->synData;

	for (int type = tInfo->startTypeThread; type < tInfo->endTypeThread; type++) {

		int globalActListSize = sharedData->hList[type][0].synapseListSize * sharedData->hList[type][0].activationListSize * tInfo->nNeurons[type];
		hipMemcpy(synData->activationListGlobal[type], sharedData->hList[type][0].activationList,
				sizeof(ftype) * globalActListSize, hipMemcpyDeviceToHost);

		SynapticChannels *synChannel = sharedData->matrixList[type][0].synapticChannels;
		hipMemcpy(synData->activationListPosGlobal[type], sharedData->hList[type][0].activationListPos,
				sizeof(ucomp) * synChannel->synapseListSize * tInfo->nNeurons[type], hipMemcpyDeviceToHost);

	}

}

void GpuSimulationControl::transferHinesStructToGpu() {

	for(int type = tInfo->startTypeThread;type < tInfo->endTypeThread;type++){

		hipMalloc((void**)((((&(sharedData->hGpu[type]))))), sizeof (HinesStruct) * tInfo->nNeurons[type]);
		hipMemcpy(sharedData->hGpu[type], sharedData->hList[type], sizeof (HinesStruct) * tInfo->nNeurons[type], hipMemcpyHostToDevice);
		checkCUDAError("Memory Allocation [hGPU]:");
	}
}


int GpuSimulationControl::prepareExecution(int type) {

	int nNeurons = tInfo->nNeurons[type];
	int nKernelSteps = kernelInfo->nKernelSteps;

	HinesStruct **hListPtr = &(sharedData->hList[type]);

	HinesStruct *hList = (HinesStruct *)malloc(nNeurons*sizeof(HinesStruct)); //new HinesStruct[nNeurons];

	HinesMatrix & m0 = sharedData->matrixList[type][0];
	int nComp = m0.nComp;
	int nCompActive = m0.activeChannels->getCompListSize();
	int nSynaptic = m0.synapticChannels->synapseListSize;

	/******************************************************************************************
	 * Allocates the ftype memory for all neurons and copies data to device
	 *******************************************************************************************/
	int fSharedMemMatrixSize    = sizeof(ftype) * (3*nComp + m0.mulListSize + m0.leftListSize); //+ nComp*nComp;
	int fSharedMemSynapticSize  = sizeof(ftype) * nSynaptic * SYN_CONST_N;
	int fSharedMemSize = fSharedMemMatrixSize + fSharedMemSynapticSize;

	int fExclusiveMemMatrixSize   = sizeof(ftype) * (5*nComp + m0.leftListSize);
	int fExclusiveMemActiveSize   = sizeof(ftype) * m0.activeChannels->ftypeMemSize;
	int fExclusiveMemSynapticSize = sizeof(ftype) * nSynaptic * (SYN_STATE_N + m0.synapticChannels->activationListSize);
	int fExclusiveMemSize = fExclusiveMemMatrixSize + fExclusiveMemActiveSize + fExclusiveMemSynapticSize + sizeof(ftype)*nComp*nKernelSteps;

	ftype *fMemory;
	hipMalloc((void **)(&(fMemory)), fSharedMemSize + fExclusiveMemSize * nNeurons);
	ftype *fSharedMemMatrixAddress = m0.Cm;
	ftype *fSharedMemSynapticAddress = m0.synapticChannels->synConstants;

	hipMemcpy(fMemory, fSharedMemMatrixAddress, fSharedMemMatrixSize, hipMemcpyHostToDevice);
	hipMemcpy(fMemory+fSharedMemMatrixSize/sizeof(ftype), fSharedMemSynapticAddress, fSharedMemSynapticSize, hipMemcpyHostToDevice);


	/******************************************************************************************
	 * Allocates the ucomp memory for all neurons and copies data to device
	 *******************************************************************************************/

	int uExclusiveMemSynapticSize = sizeof(ucomp) * nSynaptic;
	ucomp *uExclusiveMemSynaptic;
	hipMalloc((void **)(&(uExclusiveMemSynaptic)), uExclusiveMemSynapticSize * nNeurons);

	int uMemMatrixSize    = sizeof(ucomp) * ((m0.mulListSize + m0.leftListSize) * 2 + nComp);
	int uMemActiveSize    = sizeof(ucomp) * m0.activeChannels->ucompMemSize;
	int uMemSynapticSize  = sizeof(ucomp) * 2 * nSynaptic;
	int uMemSize = uMemMatrixSize + uMemActiveSize + uMemSynapticSize;

	ucomp *uMemory;
	hipMalloc((void **)(&(uMemory)), uMemSize);
	ucomp *uMemActiveAddress   = uMemory + uMemMatrixSize / sizeof(ucomp);
	ucomp *uMemSynapticAddress = uMemActiveAddress + uMemActiveSize / sizeof(ucomp);

	hipMemcpy(uMemory,             m0.ucompMemory,                       uMemMatrixSize,   hipMemcpyHostToDevice);
	hipMemcpy(uMemActiveAddress,   m0.activeChannels->ucompMem,          uMemActiveSize,   hipMemcpyHostToDevice);
	hipMemcpy(uMemSynapticAddress, m0.synapticChannels->synapseCompList, uMemSynapticSize, hipMemcpyHostToDevice);

	//hipMemcpy(uMemSynapticAddress, m0.synapticChannels->synapseCompList, uMemSynapticSize, hipMemcpyHostToDevice);
	//hipMemcpy(uMemActiveAddress,   m0.activeChannels->getCompList(),     uMemActiveSize,   hipMemcpyHostToDevice); // TODO: old active

	printf("Memory for Neurons: %10.3f MB for %d neurons of type %d.\n",(fSharedMemSize + uMemSize + (fExclusiveMemSize +  uExclusiveMemSynapticSize) * nNeurons)/(1024.*1024.), nNeurons, type);

	/******************************************************************************************
	 * Prepare the MatrixStruct h for each neuron in the GPU
	 *******************************************************************************************/
	for (int neuron = 0; neuron < nNeurons; neuron++ ) {

		HinesMatrix & m = sharedData->matrixList[type][neuron];
		HinesStruct & h = hList[neuron];

		/****************************************************
		 * Fields of the HinesStruct
		 ****************************************************/
		h.currStep = m.currStep;
		h.vRest = m.vRest;
		h.dx = m.dx;
		h.nComp = m.nComp;
		h.dt = m.dt;
		h.triangAll = m.triangAll;
		h.mulListSize = m.mulListSize;
		h.leftListSize = m.leftListSize;
		h.type = type;
		h.nNeurons = nNeurons;

		/****************************************************
		 * ftype memory shared among all neurons
		 ****************************************************/
		h.memoryS = fMemory;
		h.Cm = h.memoryS;
		h.Ra = h.Cm + nComp;
		h.Rm = h.Ra + nComp;
		h.leftList = h.Rm + nComp;
		h.mulList  = h.leftList + m.leftListSize; // Used only when triangAll = 0

		/****************************************************
		 * ftype memory allocated per neuron
		 ****************************************************/
		h.memoryE = fMemory + fSharedMemSize/sizeof(ftype) + neuron*fExclusiveMemSize/sizeof(ftype);
		ftype *exclusiveAddressM = m.rhsM;
		hipMemcpy(h.memoryE, exclusiveAddressM, fExclusiveMemMatrixSize, hipMemcpyHostToDevice);
		// must match the order in HinesMatrix.cpp
		h.rhsM = h.memoryE	;
		h.vmList = h.rhsM + nComp;
		h.vmTmp = h.vmList + nComp;
		h.curr = h.vmTmp + nComp;
		h.active = h.curr + nComp;
		h.triangList = h.active + nComp; // triangularized list
		h.vmTimeSerie = h.triangList + m.leftListSize;

		/****************************************************
		 * ucomp memory shared among all neurons
		 ****************************************************/
		h.mulListComp    = uMemory;
		h.mulListDest    = h.mulListComp  + h.mulListSize;
		h.leftListLine   = h.mulListDest  + h.mulListSize;
		h.leftListColumn = h.leftListLine + h.leftListSize;
		h.leftStartPos   = h.leftListColumn + h.leftListSize;


		/****************************************************
		 * Active channels using the old  and new implementations
		 ****************************************************/
		if (nCompActive > 0 && m.activeChannels->channelInfo == 0) {

		}
		else if (m.activeChannels->channelInfo != 0) {

			ftype *activeMemAddress = h.vmTimeSerie + nComp*nKernelSteps;
			hipMemcpy(activeMemAddress, m.activeChannels->ftypeMem, fExclusiveMemActiveSize, hipMemcpyHostToDevice);

			h.nChannels    = m.activeChannels->nChannels;
			h.compListSize = m.activeChannels->nActiveComp;
			h.nGatesTotal  = m.activeChannels->nGatesTotal;

			h.channelEk   = activeMemAddress;
			h.channelGbar = h.channelEk   + h.nChannels;
			h.eLeak       = h.channelGbar + h.nChannels;
			h.gActive	  = h.eLeak       + h.compListSize;
			h.gateState   = h.gActive	  + h.compListSize;
			h.gatePar     = h.gateState   + h.nGatesTotal;


			h.compList    = uMemActiveAddress;
			h.channelInfo = h.compList    + h.compListSize;
			h.gateInfo    = h.channelInfo + (h.nChannels * N_CHANNEL_FIELDS);
		}
		checkCUDAError("Memory Allocation after active:");

		/****************************************************
		 * Synaptic Channels
		 ****************************************************/
		if (m.synapticChannels != 0) {

			SynapticChannels *synChan = m.synapticChannels;
			h.synapseListSize    = synChan->synapseListSize;
			h.activationListSize = synChan->activationListSize;

			h.synapseCompList = uMemSynapticAddress;
			h.synapseTypeList = h.synapseCompList + h.synapseListSize;

			h.synConstants = fMemory + fSharedMemMatrixSize/sizeof(ftype);

			// exclusive fmemory
			h.synState = h.gatePar + h.nGatesTotal * N_GATE_FUNC_PAR;
			hipMemcpy(h.synState, synChan->synState,
					sizeof(ftype) * h.synapseListSize * SYN_STATE_N, hipMemcpyHostToDevice);

			h.activationList = h.synState + h.synapseListSize * SYN_STATE_N; // h.synapseListSize * h.activationListSize;
			// TODO: should not be necessary
			hipMemcpy(h.activationList, synChan->activationList,
					sizeof(ftype) * h.synapseListSize * h.activationListSize, hipMemcpyHostToDevice);

			h.activationListPos = uExclusiveMemSynaptic + neuron * uExclusiveMemSynapticSize/sizeof(ucomp);
			hipMemcpy(h.activationListPos, synChan->activationListPos, uExclusiveMemSynapticSize, hipMemcpyHostToDevice);

			// Used for spike generation
			h.lastSpike 		= m.lastSpike;
			h.spikeTimeListSize = m.spikeTimeListSize;
			h.threshold         = m.threshold;
			h.minSpikeInterval  = m.minSpikeInterval;
		}
		checkCUDAError("Memory Allocation after synaptic:");

		//if (benchConf.simCommMode == NN_GPU)
		sharedData->matrixList[type][neuron].freeMem();
	}

	*hListPtr = hList;

	return 0;
}

void GpuSimulationControl::performGPUCommunications(int type, struct RandomSpikeInfo & randomSpkInfo) {

	/**
	 * Transfers the list of random spikes to the GPU
	 */
	for(int i = randomSpkInfo.nRandom;i < randomSpkInfo.listSize;i++){
		randomSpkInfo.spikeTimes[i] = -1;
		randomSpkInfo.spikeDest[i] = -1;
	}

	SynapticData *synData = sharedData->synData;
	int threadNumber = tInfo->threadNumber;
	int *nBlocksComm  = kernelInfo->nBlocksComm;
	int *nThreadsComm = kernelInfo->nThreadsComm;

	ftype *randomSpikeTimesDev;
	int *randomSpikeDestDev;
	hipMalloc((void**)(&randomSpikeTimesDev), sizeof (ftype) * randomSpkInfo.listSize);
	hipMalloc((void**)(&randomSpikeDestDev), sizeof (int) * randomSpkInfo.listSize);
	hipMemcpy(randomSpikeTimesDev, randomSpkInfo.spikeTimes, sizeof (ftype) * randomSpkInfo.listSize, hipMemcpyHostToDevice);
	hipMemcpy(randomSpikeDestDev,  randomSpkInfo.spikeDest,  sizeof (int)   * randomSpkInfo.listSize, hipMemcpyHostToDevice);



	uint64 connTmp = 0;
	if (threadNumber == 0 && benchConf.gpuCommBenchMode == GPU_COMM_SIMPLE)
		connTmp = gettimeInMilli();

	/**
	 * TODO: Remove Me [MPI]
	 * Used only during debugging to check the number of received spikes per process
	 */
//	int *nReceivedSpikesDev0, *nReceivedSpikesDev1, *nReceivedSpikesHost0, *nReceivedSpikesHost1;
//	hipMalloc ( (void **) &nReceivedSpikesDev0, sizeof(int) * nBlocksComm[type]);
//	hipMalloc ( (void **) &nReceivedSpikesDev1, sizeof(int) * nBlocksComm[type]);
//	nReceivedSpikesHost0 = (int *)malloc(sizeof(int) * nBlocksComm[type]);
//	nReceivedSpikesHost1 = (int *)malloc(sizeof(int) * nBlocksComm[type]);


	performCommunicationsG <<<nBlocksComm[type], nThreadsComm[type], kernelInfo->sharedMemSizeComm>>>(
			tInfo->nNeurons[type], sharedData->connGpuListDevice[type],
			synData->nGeneratedSpikesGpusDev[threadNumber], synData->genSpikeTimeListGpusDev[threadNumber],
			sharedData->hGpu[type], randomSpikeTimesDev, randomSpikeDestDev, randomSpkInfo.nRandom);

	/**
	 * TODO: Remove Me [MPI]
	 * Used only during debugging to check the number of received spikes per process
	 */
//	hipMemcpy(nReceivedSpikesHost0, nReceivedSpikesDev0, sizeof(int) * nBlocksComm[type], hipMemcpyDeviceToHost);
//	hipMemcpy(nReceivedSpikesHost1, nReceivedSpikesDev1, sizeof(int) * nBlocksComm[type], hipMemcpyDeviceToHost);
//	long nReceived0 = 0, nReceived1 = 0;
//	for (int k=0; k < nBlocksComm[type]; k++ ) {
//		nReceived0 += nReceivedSpikesHost0[k];
//		nReceived1 += nReceivedSpikesHost1[k];
//	}
	//printf("nReceived=0:%5ld|1:%5ld for type %d\n", nReceived0, nReceived1, type);

//	hipFree ( nReceivedSpikesDev0);
//	hipFree ( nReceivedSpikesDev1);
//	free(nReceivedSpikesHost0);
//	free(nReceivedSpikesHost1);

	hipDeviceSynchronize();
	checkCUDAError("After performCommunicationsG Kernel2:");

	if (threadNumber == 0 && benchConf.gpuCommBenchMode == GPU_COMM_SIMPLE) {
		hipDeviceSynchronize();
		bench.connRead += (gettimeInMilli()-connTmp);
	}



	//int **countReceivedSpikesCpu(synData->v, nNeurons[type], nBlocksComm[type], synData->nGeneratedSpikesHost);
	//hipDeviceSynchronize();

	hipFree(randomSpikeTimesDev);
	hipFree(randomSpikeDestDev);
	//hipDeviceSynchronize();
}

void GpuSimulationControl::performGpuNeuronalProcessing() {

	for (int type = tInfo->startTypeThread; type < tInfo->endTypeThread; type++) {

		int nThreadsProc =  tInfo->nNeurons[type]/kernelInfo->nBlocksProc[type];
		if (tInfo->nNeurons[type] % kernelInfo->nBlocksProc[type]) nThreadsProc++;

		//printf("launching kernel for type %d...\n", type);
		SynapticData *synData = sharedData->synData;

		solveMatrixG<<<kernelInfo->nBlocksProc[type], nThreadsProc, kernelInfo->sharedMemSizeProc>>>(
				sharedData->hGpu[type], kernelInfo->nKernelSteps, tInfo->nNeurons[type], synData->vmListDevice[type]);

		checkCUDAError("After SolveMatrixG Kernel:");
	}

}

void GpuSimulationControl::checkVmValues()
{
	SynapticData *synData = sharedData->synData;
    for (int type = tInfo->startTypeThread; type < tInfo->endTypeThread; type++)
					for (int n = 0; n < tInfo->nNeurons[type]; n++)
						if ( synData->vmListHost[type][n] < -500 || 500 < synData->vmListHost[type][n] || synData->vmListHost[type][n] == 0.000000000000) {
							printf("********* type=%d neuron=%d %.2f\n", type, n, synData->vmListHost[type][n]);
							assert(false);
						}
}



void GpuSimulationControl::prepareGpuSpikeDeliveryStructures()
{
	SynapticData *synData = sharedData->synData;
	int threadNumber = tInfo->threadNumber;

    synData->genSpikeTimeListGpusHost[threadNumber] = (ftype**)(malloc(sizeof (ftype*) * tInfo->totalTypes));
    for (int type = 0; type < tInfo->totalTypes; type++) {
			int genSpikeTimeListSize = GENSPIKETIMELIST_SIZE;
			// The device memory for the types of the current thread are already allocated
			if (tInfo->startTypeThread <= type && type < tInfo->endTypeThread)
				synData->genSpikeTimeListGpusHost[threadNumber][type] = synData->genSpikeTimeListDevice[type];
			else
				hipMalloc ((void **) &(synData->genSpikeTimeListGpusHost[threadNumber][type]),
						sizeof(ftype) * genSpikeTimeListSize * tInfo->nNeurons[type]);
		}
    // Copies the list of pointers to the genSpikeLists of each type
    hipMalloc((void**)(&synData->genSpikeTimeListGpusDev[threadNumber]), sizeof (ftype*) * tInfo->totalTypes);
    hipMemcpy(synData->genSpikeTimeListGpusDev[threadNumber], synData->genSpikeTimeListGpusHost[threadNumber], sizeof (ftype*) * tInfo->totalTypes, hipMemcpyHostToDevice);
    synData->nGeneratedSpikesGpusHost[threadNumber] = (ucomp**)(malloc(sizeof (ucomp*) * tInfo->totalTypes));
    for(int type = 0;type < tInfo->totalTypes;type++){
        if(tInfo->startTypeThread <= type && type < tInfo->endTypeThread)
            synData->nGeneratedSpikesGpusHost[threadNumber][type] = synData->nGeneratedSpikesDevice[type];

        else
            hipMalloc((void**)(&(synData->nGeneratedSpikesGpusHost[threadNumber][type])), sizeof (ucomp) * tInfo->nNeurons[type]);

    }
    // Copies the list of pointers to the nGeneratedSpikes of each type
    hipMalloc((void**)(&synData->nGeneratedSpikesGpusDev[threadNumber]), sizeof (ucomp*) * tInfo->totalTypes);
    hipMemcpy(synData->nGeneratedSpikesGpusDev[threadNumber], synData->nGeneratedSpikesGpusHost[threadNumber], sizeof (ucomp*) * tInfo->totalTypes, hipMemcpyHostToDevice);
}

void GpuSimulationControl::createGpuCommunicationStructures()
{

	int *nBlocksComm = kernelInfo->nBlocksComm;

    for(int destType = tInfo->startTypeThread;destType < tInfo->endTypeThread;destType++){
        sharedData->connGpuListHost[destType] =
        		createGpuConnections(sharedData->connInfo, destType, tInfo->nNeurons, nBlocksComm[destType]);

        hipMalloc((void**)(&(sharedData->connGpuListDevice[destType])), nBlocksComm[destType] * sizeof (ConnGpu));
        hipMemcpy(sharedData->connGpuListDevice[destType], sharedData->connGpuListHost[destType], nBlocksComm[destType] * sizeof (ConnGpu), hipMemcpyHostToDevice);

        kernelInfo->nThreadsComm[destType] = sharedData->connGpuListHost[destType][0].nNeuronsGroup;
    }
}

void GpuSimulationControl::copyGeneratedSpikeListsToGPU()
{

	// Consider that each thread has its own device
	for (int type=0; type < tInfo->totalTypes; type++) {
		if (type < tInfo->startTypeThread || tInfo->endTypeThread <= type) {

			hipMemcpy(sharedData->synData->genSpikeTimeListGpusHost[tInfo->threadNumber][type],
					sharedData->synData->genSpikeTimeListHost[type],
					sizeof(ftype) * GENSPIKETIMELIST_SIZE *tInfo-> nNeurons[type], hipMemcpyHostToDevice);

			hipMemcpy(sharedData->synData->nGeneratedSpikesGpusHost[tInfo->threadNumber][type],
					sharedData->synData->nGeneratedSpikesHost[type],
					sizeof(ucomp) * tInfo->nNeurons[type], hipMemcpyHostToDevice);
		}
	}
}

void GpuSimulationControl::readGeneratedSpikesFromGPU()
{
    /*--------------------------------------------------------------
     * Reads information from spike sources
     *--------------------------------------------------------------*/
    if(benchConf.verbose == 1)
        printf("Getting spikes %d\n", tInfo->threadNumber);

    for(int type = tInfo->startTypeThread;type < tInfo->endTypeThread;type++){

        hipMemcpy(sharedData->synData->genSpikeTimeListHost[type], sharedData->synData->genSpikeTimeListDevice[type],
        		sizeof (ftype) * sharedData->hList[type][0].spikeTimeListSize * tInfo->nNeurons[type], hipMemcpyDeviceToHost);

        hipMemcpy(sharedData->synData->nGeneratedSpikesHost[type], sharedData->synData->nGeneratedSpikesDevice[type],
        		sizeof (ucomp) * tInfo->nNeurons[type], hipMemcpyDeviceToHost);

        checkCUDAError("Synapses2:");
    }

}

void GpuSimulationControl::configureGpuKernel()
{

    /*--------------------------------------------------------------
	 * Select the device attributed to each thread
	 *--------------------------------------------------------------*/
    int nDevices = 0;
    hipGetDeviceCount(&nDevices);
    hipSetDevice((tInfo->threadNumber + tInfo->currProcess + 1) % nDevices);
    hipGetDevice(&(tInfo->deviceNumber));
    tInfo->prop = new struct hipDeviceProp_t;
    hipGetDeviceProperties(tInfo->prop, tInfo->deviceNumber);

    checkCUDAError("Device selection:");
    //--------------------------------------------------------------
    /*--------------------------------------------------------------
	 * Configure number of threads and shared memory size for each kernel
	 *--------------------------------------------------------------*/
    kernelInfo->maxThreadsProc = 64;
    kernelInfo->maxThreadsComm = 16;
    kernelInfo->sharedMemSizeProc = 15 * 1024; // Valid for capability 1.x (16kB)
    kernelInfo->sharedMemSizeComm = 15 * 1024; // Valid for capability 1.x (16kB)
    if(tInfo->prop->major == 2){
        kernelInfo->maxThreadsProc = 256;
        kernelInfo->maxThreadsComm = 32; // or can be 64
        kernelInfo->sharedMemSizeProc = 47 * 1024; // Valid for capability 2.x (48kB)
        kernelInfo->sharedMemSizeComm = 15 * 1024; // Valid for capability 2.x (48kB)
    }
    else if(tInfo->prop->major == 3){
      kernelInfo->maxThreadsProc = 384;//384;
      kernelInfo->maxThreadsComm = 32; // or can be 64
      kernelInfo->sharedMemSizeProc = 47 * 1024; // Valid for capability 2.x (48kB)
      kernelInfo->sharedMemSizeComm = 15 * 1024; // Valid for capability 2.x (48kB)
    }
    //--------------------------------------------------------------
    for(int type = tInfo->startTypeThread;type < tInfo->endTypeThread;type++){
        // Number of blocks: multiple of #GPU multiprocessors and respects maxThreadsProc condition
        kernelInfo->nBlocksProc[type] = tInfo->prop->multiProcessorCount * (tInfo->nNeurons[type] / kernelInfo->maxThreadsProc / tInfo->prop->multiProcessorCount);
        if(tInfo->nNeurons[type] % kernelInfo->maxThreadsProc != 0 || (tInfo->nNeurons[type] / kernelInfo->maxThreadsProc) % kernelInfo->maxThreadsProc != 0)
            kernelInfo->nBlocksProc[type] += tInfo->prop->multiProcessorCount;

    }

    for(int destType = tInfo->startTypeThread;destType < tInfo->endTypeThread;destType++){

    	// Number of blocks: multiple of #GPU multiprocessors and respects maxThreadsComm condition
    	kernelInfo->nBlocksComm[destType] = tInfo->prop->multiProcessorCount * (tInfo->nNeurons[destType] / kernelInfo->maxThreadsComm / tInfo->prop->multiProcessorCount);
        if(tInfo->nNeurons[destType] % kernelInfo->maxThreadsComm != 0 || (tInfo->nNeurons[destType] / kernelInfo->maxThreadsComm) % kernelInfo->maxThreadsComm != 0)
        	kernelInfo->nBlocksComm[destType] += tInfo->prop->multiProcessorCount;
    }

    //	if (nComp0 <= 4) nThreads = (sizeof (ftype) == 4) ? 196 : 96;
    //	else if (nComp0 <= 8) nThreads = (sizeof (ftype) == 4) ? 128 : 64;
    //	else if (nComp0 <= 12) nThreads = (sizeof (ftype) == 4) ? 96 : 32;
    //	else if (nComp0 <= 16) nThreads = (sizeof (ftype) == 4) ? 64 : 32;

}

void GpuSimulationControl::updateSharedDataInfo()
{
    sharedData->hList = new HinesStruct*[tInfo->totalTypes];
    sharedData->hGpu = new HinesStruct*[tInfo->totalTypes];
    sharedData->synData = new SynapticData;
    sharedData->synData->activationListGlobal = 0;
    sharedData->synData->totalTypes = tInfo->totalTypes;
    sharedData->neuronInfoWriter = new NeuronInfoWriter(tInfo);
    sharedData->spkStat = new SpikeStatistics(tInfo->nNeurons, tInfo->nTypes, tInfo->totalTypes, tInfo->sharedData->typeList, tInfo->startTypeProcess, tInfo->endTypeProcess);
    kernelInfo->nThreadsComm = new int[tInfo->totalTypes];
    kernelInfo->nBlocksComm = new int[tInfo->totalTypes];
    kernelInfo->nBlocksProc = new int[tInfo->totalTypes];
}



void GpuSimulationControl::addToInterleavedSynapticActivationList(
		ftype *activationList, ucomp *activationListPos, int activationListSize, int neuron, int nNeurons,
		ftype currTime, ftype dt, ucomp synapse, ftype spikeTime, ftype delay, ftype weight) {

	//pthread_mutex_lock (&addSpikeMutex);

	ftype fpos = (spikeTime + delay - currTime) / dt;

	ucomp pos  = ( activationListPos[synapse] + (ucomp)fpos + 1 ) % activationListSize;
	pos       += synapse * activationListSize;

	ucomp nextPos  = ( pos + 1 ) % activationListSize;
	nextPos       += synapse * activationListSize;

	ftype diff = fpos - (int)fpos;

	// TODO: race conditions can occur here with multiple threads
	activationList[    pos * nNeurons + neuron] += (weight / dt) * ( 1 - diff );
	activationList[nextPos * nNeurons + neuron] += (weight / dt) * diff;

//	activationList[    pos * nNeurons + neuron] = (weight / dt) * ( 1 - diff );
//	activationList[nextPos * nNeurons + neuron] = (weight / dt) * diff;

	//pthread_mutex_unlock (&addSpikeMutex);
}
