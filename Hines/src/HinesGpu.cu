#include "hip/hip_runtime.h"
/*
 * HinesGpu.cu
 *
 *  Created on: 06/06/2009
 *      Author: rcamargo
 */

/**
 * TODO: otimizar
 * - acessar sTriangList com POS
 * - Ver acessos à memória global
 */

/**
 * TODO:
 * - Otimizar (usar memória compartilhada)
 * - Otimizar (reduzir número de alocações de memória para copiar lista de spikes)
 */

//extern "C" {
#include "HinesMatrix.hpp"
#include "PlatformFunctions.hpp"
#include "HinesStruct.hpp"
#include <cassert>

#include <hip/hip_runtime.h> // Necessary to allow better eclipse integration
#include <hip/hip_runtime_api.h> // Necessary to allow better eclipse integration
#include <> // Necessary to allow better eclipse integration
#include <hip/device_functions.h> // Necessary to allow better eclipse integration

//#define POS(i) (i) + nComp*threadIdx.x
#define POS(i) (i)*blockDim.x+threadIdx.x
//#define POS1(i) (i) + leftListSize*threadIdx.x
#define POS1(i) (i)*blockDim.x + threadIdx.x

#define NLOCALSPIKES 16 // 16

/***************************************************************************
 * This part is executed in every integration step
 ***************************************************************************/

__device__ void evaluateSynapticCurrentsNew( HinesStruct *hList, ftype *active, ftype *vmList,
		ftype currTime,	int synapseListSize, ftype *synConstants, ftype *synState, ucomp *synapseCompList,
		int activationListSize, ftype *activationList, ucomp *activationListPos ) {

	int neuron = blockIdx.x * blockDim.x + threadIdx.x;

	for (int syn=0; syn < synapseListSize; syn++) {
		int currPos = (syn * activationListSize) + activationListPos[syn];
		currPos = currPos * hList[neuron].nNeurons + neuron; // global interleave lList
		ftype activation = activationList[ currPos ];
		activationList[ currPos ] = 0;

		activationListPos[syn] = (activationListPos[syn] + 1) % activationListSize;

		int synComp = synapseCompList[syn];

		// TODO: problem is in one of the lines below
		synState[SYN_STATE_X] = synConstants[SYN_MOD] * activation * synConstants[SYN_X1] + synState[SYN_STATE_X] * synConstants[SYN_X2];
		synState[SYN_STATE_Y] = synState[SYN_STATE_X] * synConstants[SYN_Y1] + synState[SYN_STATE_Y] * synConstants[SYN_Y2];

		ftype gsyn = synState[SYN_STATE_Y] * synConstants[SYN_NORM];

		active[POS(synComp)] += (vmList[POS(synComp)] - synConstants[SYN_EK]) * gsyn;

		synState     += SYN_STATE_N;
		synConstants += SYN_CONST_N;
	}

	synState     -= SYN_STATE_N * synapseListSize;
	synConstants -= SYN_CONST_N * synapseListSize;
}


/**
 * Find the gate openings in the next time step
 * m(t + dt) = a + b m(t - dt)
 */
__device__ void evaluateGatesGNew( HinesStruct *hList, ftype *vmListLocal, int nChannels,
		ucomp *channelInfo, ftype *gatePar, ucomp *gateInfo, ftype *gateState) {

	HinesStruct & h = hList[blockIdx.x * blockDim.x + threadIdx.x];

	ftype alpha, beta, a, b;
	ftype V;
	ftype dtRev = 1/h.dt;

	int pos=0;
	for (int ch=0; ch<nChannels; ch++) {

		int nGates = channelInfo[ch*N_CHANNEL_FIELDS + CH_NGATES];
		//V = vmList[ channelInfo[ch*N_CHANNEL_FIELDS + CH_COMP] ];
		V = vmListLocal[ POS( channelInfo[ch*N_CHANNEL_FIELDS + CH_COMP] ) ];

		for (int gt=0; gt < nGates; gt++, pos++) {

            // (EXPONENTIAL): alpha(v) = A exp((v-V0)/B)
            // (SIGMOID):     alpha(v) = A / (exp((v-V0)/B) + 1)
            // (LINOID):      alpha(v) = A (v-V0) / (exp((v-V0)/B) - 1)

			// alpha_function
			ftype v0 = gatePar[A_V0];
			switch( gateInfo[pos * N_GATE_FIELDS + ALPHA_FUNCTION] ) {
			case EXPONENTIAL:
				alpha = gatePar[A_A] * expf((V-v0)/gatePar[A_B]);
				break;
			case SIGMOID:
				alpha = gatePar[A_A] / ( expf( (V-v0)/gatePar[A_B] ) + 1);
				break;
			case LINOID:
				alpha = (V != v0) ? gatePar[A_A] * (V-v0) / (expf((V-v0)/gatePar[A_B]) - 1) : gatePar[A_A] * gatePar[A_B];
				break;
			}

			// beta_function
			v0 = gatePar[B_V0];
			switch( gateInfo[pos * N_GATE_FIELDS + BETA_FUNCTION] ) {
			case EXPONENTIAL:
				beta = gatePar[B_A] * expf((V-v0)/gatePar[B_B]);
				break;
			case SIGMOID:
				beta = gatePar[B_A] / ( expf( (V-v0)/gatePar[B_B] ) + 1);
				break;
			case LINOID:
				beta = (V != v0) ? gatePar[B_A] * (V-v0) / (expf((V-v0)/gatePar[B_B]) - 1) : gatePar[B_A] * gatePar[B_B];
				break;
			}

			gatePar += N_GATE_FUNC_PAR;

			a = alpha / (dtRev  + (alpha + beta)/2);
			b = (dtRev - (alpha + beta)/2) / (dtRev + (alpha + beta)/2);

			gateState[POS(pos)] = a + b * gateState[POS(pos)];
		}

	}
}



__device__ void evaluateCurrentsGNew( HinesStruct *hList, ftype *activeList, ftype *vmListLocal,  int nChannels,
		ucomp *channelInfo, ucomp *gateInfo, ftype *gateState,
		int nComp, int compListSize, ucomp *compList, ftype *eLeak) {

	//ftype *Rm, ftype *active

	int neuron = blockIdx.x * blockDim.x + threadIdx.x;
	HinesStruct & h = hList[neuron];

	evaluateGatesGNew(hList, vmListLocal, nChannels, channelInfo, h.gatePar, gateInfo, gateState);

	ftype *channelEk = h.channelEk;
	ftype *channelGbar = h.channelGbar;

	for (int i=0; i<compListSize; i++)
		h.gActive[i] = 0;

	/**
	 * Update the channel conductances
	 */
	int pos = 0;
	int actCompPos = -1;
	int lastActComp = -1;

	for (int ch=0; ch<nChannels; ch++) {

		int nGates     = channelInfo[ch*N_CHANNEL_FIELDS + CH_NGATES];
		int comp       = channelInfo[ch*N_CHANNEL_FIELDS + CH_COMP];
		ftype gChannel = channelGbar[ch];


		for (int gt=0; gt < nGates; gt++, pos++) {

//			gateState[pos]=0;

			ftype state = gateState[POS(pos)];
			switch( gateInfo[pos * N_GATE_FIELDS + GATE_POWER] ) {
			case 4:
				gChannel *= (state*state*state*state);
				break;
			case 3:
				gChannel *= (state*state*state);
				break;
			case 2:
				gChannel *= (state*state);
				break;
			case 1:
				gChannel *= state;
				break;
			default:
				gChannel *= powf(state, gateInfo[pos * N_GATE_FIELDS + GATE_POWER] );
				break;
			}

		}

		activeList[ POS(comp) ] -= gChannel * channelEk[ch] ;

		if (comp != lastActComp) {
			actCompPos++;
			lastActComp = comp;
		}
		h.gActive[ actCompPos ] += gChannel;
	}

	for (int i=0; i<compListSize; i++) {
		unsigned int comp = compList[i];
		activeList[ POS(comp) ] -=  ( 1 / h.Rm[comp] ) * ( eLeak[i] );
	}
}


__device__ void upperTriangularizeAll(HinesStruct *hList, ftype *sTriangList,
				ftype *sLeftList, ucomp *sLeftListLine, ucomp *sLeftListColumn,
				ucomp *sLeftStartPos, ftype *rhsLocal, ftype *vmListLocal,

				int nChannels, ucomp *channelInfo, ucomp *gateInfo, ftype *gateState,
				int compListSize, ucomp *compList, ftype *eLeak,

				ftype *freeMem) {

	int neuron = blockIdx.x * blockDim.x + threadIdx.x;
	HinesStruct & h = hList[neuron];

	int nComp = h.nComp;
	int leftListSize = h.leftListSize;

	ftype *active = freeMem;
	freeMem = active + blockDim.x * nComp;

	ftype *Cm = h.Cm;
	ftype *curr = h.curr;


	for (int i=0; i<nComp; i++) {
		active[ POS(i)] = 0;
	}

	//__syncthreads();

	evaluateCurrentsGNew( hList, active, vmListLocal, nChannels,
			channelInfo, gateInfo, gateState,
			nComp, compListSize, compList, eLeak);

	evaluateSynapticCurrentsNew(hList, active, vmListLocal, h.currStep * h.dt,
			h.synapseListSize, h.synConstants, h.synState, h.synapseCompList, 	//
			h.activationListSize, h.activationList, h.activationListPos);

	ftype dtRec = 1/h.dt;
	//rhsLocal[POS(0)] = (-2) * vmListLocal[POS(0)] * Cm[0] * dtRec - curr[0] + active[POS(0)];
	for (int i=0; i<nComp; i++)
		rhsLocal[POS(i)] = (-2) * vmListLocal[POS(i)] * Cm[i] * dtRec - curr[i] + active[POS(i)];

	// ***
	// 1000ms 960 16 1 -> 0.125ms
	for (int k = 0; k < leftListSize; k++)
		sTriangList[k] = sLeftList[k];

	for (int i = 0; i < h.compListSize; i++) {

		int comp = h.compList[i];
		int pos = sLeftStartPos[ comp ];

		for (; sLeftListColumn[pos] < comp && pos < leftListSize ; pos++);

		sTriangList[pos] -= h.gActive[i];
	}


	// 1000ms 960 16 1 -> 0.640ms
	for (int k = 0; k < leftListSize; k++) {

		int c = sLeftListColumn[k];
		int l = sLeftListLine[k];

		if( c < l ) {

			int pos = sLeftStartPos[c];
			for (; c == sLeftListLine[pos]; pos++)
				if (sLeftListColumn[pos] == c)
					break;

			ftype mul = -sTriangList[k] / sTriangList[pos];

			pos = sLeftStartPos[c];
			int tempK = sLeftStartPos[l];

			for (; c == sLeftListLine[pos] && pos < leftListSize; pos++) {
				for (; sLeftListColumn[tempK] < sLeftListColumn[pos] && tempK < leftListSize ; tempK++);

				sTriangList[tempK] += sTriangList[pos] * mul;
			}
			rhsLocal[POS(l)] += rhsLocal[POS(c)] * mul;
		}
	}


}


__device__ void updateRhsG(HinesStruct *hList,
						   ftype *sMulList, ucomp *sMulListComp, ucomp *sMulListDest,
						   ftype *rhsLocal, ftype *vmListLocal,

						   int nChannels, ucomp *channelInfo, ucomp *gateInfo, ftype *gateState,
						   int compListSize, ucomp *compList, ftype *eLeak,

						   ftype *freeMem) {

	int neuron = blockIdx.x * blockDim.x + threadIdx.x;
	HinesStruct & h = hList[neuron];

	int nComp = h.nComp;

	ftype *active = freeMem;
	freeMem = active + blockDim.x * nComp;

	ftype *Cm = h.Cm;
	ftype *curr = h.curr;

	for (int i=0; i<nComp; i++) {
		active[ POS(i)] = 0;
	}


	//__syncthreads();

	evaluateCurrentsGNew( hList, active, vmListLocal, nChannels,
			channelInfo, gateInfo, gateState,
			nComp, compListSize, compList, eLeak);


	evaluateSynapticCurrentsNew(hList, active, vmListLocal, h.currStep * h.dt,
			h.synapseListSize, h.synConstants, h.synState, h.synapseCompList, 	//
			h.activationListSize, h.activationList, h.activationListPos);

	ftype dtRec = 1/h.dt;
	for (int i=0; i<nComp; i++)
		rhsLocal[POS(i)] = (-2) * vmListLocal[POS(i)] * Cm[i] * dtRec - curr[i] + active[POS(i)];

	int mulListSize = h.mulListSize;
	for (int mulListPos = 0; mulListPos < mulListSize; mulListPos++) {
		int dest = sMulListDest[mulListPos];
		int pos  = sMulListComp[mulListPos];
		rhsLocal[POS(dest)] += rhsLocal[POS(pos)] * sMulList[mulListPos];
	}

}

__device__ void backSubstituteG(HinesStruct *hList, 
								ftype *sTriangList, ucomp *sLeftListLine, ucomp *sLeftListColumn, 
								ftype *rhsLocal, ftype *vmListLocal, ftype* freeMem) {

	int neuron = blockIdx.x * blockDim.x + threadIdx.x;
	HinesStruct & h = hList[neuron];

	ucomp nComp = h.nComp;
	int leftListSize = h.leftListSize;

	ftype *vmTmpLocal = freeMem;

	if (h.triangAll == 0 && h.compListSize > 0) // has active channels only in soma
		vmTmpLocal[POS(nComp-1)] = rhsLocal[POS(nComp-1)] / ( sTriangList[(leftListSize-1)] - h.gActive[0]);
		//vmTmpLocal[POS(nComp-1)] = rhsLocal[POS(nComp-1)] / ( sTriangList[(leftListSize-1)] - h.gNaChannel[0] - h.gKChannel[0] );
	else
		vmTmpLocal[POS(nComp-1)] = rhsLocal[POS(nComp-1)] / sTriangList[(leftListSize-1)];


	ftype tmp = 0;
	for (int leftListPos = leftListSize-2; leftListPos >=0 ; leftListPos--) {
		ucomp line   = sLeftListLine[(leftListPos)];
		ucomp column = sLeftListColumn[(leftListPos)];
		if (line == column) {
			vmTmpLocal[POS(line)] = (rhsLocal[POS(line)] - tmp) * (1 / sTriangList[(leftListPos)]);
			tmp = 0;
		}
		else
			tmp += vmTmpLocal[POS(column)] * sTriangList[(leftListPos)];
	}

	for (int l = 0 ; l < nComp; l++)
		vmListLocal[POS(l)] = 2 * vmTmpLocal[POS(l)] - vmListLocal[POS(l)];

	//if (h.type == 0 && neuron == 1) printf("vmList=%.4f\n", vmListLocal[POS(0)]);

}

__global__ void solveMatrixG(HinesStruct *hList, int nSteps, int nNeurons, ftype *vmListGlobal) {


	//return;

	//printf("Starting Kernel...\n");

	int neuron = blockIdx.x * blockDim.x + threadIdx.x;
	if (neuron >= nNeurons) return;
	HinesStruct & h = hList[neuron];
	ucomp nComp = h.nComp;
	ucomp triangAll = h.triangAll;
	ucomp nGatesTotal = h.nGatesTotal;

	/******************************************************************************************
	 * Alocates the shared memory
	 *******************************************************************************************/

	// (ftype * 5 + ucomp * 10) * nComp
	// ftype=4 e ucomp=2 e ncomp = 8   ->  320 bytes 
	// ftype=4 e ucomp=2 e ncomp = 64  -> 2560 bytes  
	extern __shared__ ftype sharedMem[]; 
	ftype *sLeftList       = (ftype *)sharedMem;
	ucomp *sLeftListLine   = (ucomp *)&(sLeftList[h.leftListSize]); 	
	ucomp *sLeftListColumn = (ucomp *)&(sLeftListLine[h.leftListSize]); 

	ftype *sMulList     = (ftype *)&(sLeftListColumn[h.leftListSize]); // mulSize is zero when triangAll is 1
	ucomp *sMulListComp = (ucomp *)&(sMulList[h.mulListSize]); 	
	ucomp *sMulListDest = (ucomp *)&(sMulListComp[h.mulListSize]); 

	ucomp *sLeftStartPos = (ucomp *)&(sMulListDest[h.mulListSize]);

	ucomp *sActiveCompList = &(sLeftStartPos[nComp]); // No significant speedup
	ucomp *sChannelInfo    = &(sActiveCompList[h.compListSize]); // No significant speedup
	ucomp *sGateInfo       = &(sChannelInfo[h.nChannels * N_CHANNEL_FIELDS]); // small speedup

//	int nChannelTypes = h.nChannelTypes;
//	ftype *sTau		= (ftype *)&(sGateInfo[nGatesTotal * N_GATE_FIELDS]);
//	ftype *sGmax 	= (ftype *)&(sTau[nChannelTypes*2]);
//	ftype *sEsyn 	= (ftype *)&(sGmax[nChannelTypes]);

	ftype *lastSharedAddress = (ftype *)&(sGateInfo[nGatesTotal * N_GATE_FIELDS]);

	/******************************************************************************************
	 * Allocate for each individual neuron
	 *******************************************************************************************/

	// nThreads * nComp * ftype * 2
	// 32 * [8 ] * 4 * 2 = 32 * 64 = 2K
	// 32 * [32] * 4 * 2 =         = 8K

	ftype *rhsLocal = (ftype *)lastSharedAddress;
	ftype *vmListLocal = rhsLocal + blockDim.x * nComp;

//	ftype *sChannelEk   = vmListLocal  + blockDim.x * nComp;
//	ftype *sChannelGbar = sChannelEk   + blockDim.x * h.nChannels;
//	ftype *sELeak       = sChannelGbar + blockDim.x * h.nChannels;
//	ftype *sGatePar     = sGateState   + blockDim.x * (h.gatePar - h.gateState);

	ftype *sGateState   = vmListLocal + blockDim.x * nComp; //sELeak + blockDim.x * h.compListSize;
	ftype *freeMem = sGateState + blockDim.x * nGatesTotal;
	ftype *sTriangList = 0;

	if (triangAll == 1) {
		sTriangList = freeMem + threadIdx.x * h.leftListSize;
		freeMem  = freeMem + blockDim.x * h.leftListSize;
	}

	/******************************************************************************************
	 * Initializaes the shared memory
	 *******************************************************************************************/

//	for (int id=0; id < nChannelTypes; id++ ) {
//		sTau[2*id]   = h.tau[2*id];
//		sTau[2*id+1] = h.tau[2*id+1];
//		sGmax[id] 	 = h.gmax[id];
//		sEsyn[id] 	 = h.esyn[id];
//
//	}

	for (int k=0; k < nComp; k ++ )
		sLeftStartPos[k] = h.leftStartPos[k];

	for (int k=0; k < nGatesTotal; k ++ )
		sGateState[POS(k)] = h.gateState[k];

	for (int i=nGatesTotal*N_GATE_FIELDS-1; i >=0; i--)
		sGateInfo[i] = h.gateInfo[i];

	for (int i=h.nChannels*N_CHANNEL_FIELDS-1; i >= 0 ; i--)
		sChannelInfo[i] = h.channelInfo[i];

	for (int i=h.compListSize-1; i >= 0 ; i--)
		sActiveCompList[i] = h.compList[i];

	for (int k=0; k < h.leftListSize; k ++ ) {
		if (triangAll == 0) sLeftList[k] = h.triangList[k];
		else				sLeftList[k] = h.leftList[k];
		sLeftListLine[k]   = h.leftListLine[k];
		sLeftListColumn[k] = h.leftListColumn[k];
	}

	if (triangAll == 0) {
		for (int k=0; k < h.mulListSize; k ++ ) {
			sMulList[k]     = h.mulList[k];
			sMulListComp[k] = h.mulListComp[k];
			sMulListDest[k] = h.mulListDest[k];
		}
	}

	for (int k=0; k < nComp; k++ )
		vmListLocal[POS(k)] = h.vmList[k];

//	for (int k=0; k < h.compListSize; k++ ) {
//		nGate[POS(k)] = h.n[k];
//		hGate[POS(k)] = h.h[k];
//		mGate[POS(k)] = h.m[k];
//	}

	/******************************************************************************************
	 * Perform the simulation
	 *******************************************************************************************/

	ftype dt = h.dt;
	int currStep = h.currStep;
	ucomp nGeneratedSpikes = 0;

	for(int gStep = 0; gStep < nSteps; gStep++ ) {

		//printf ("SolveMatrixG: Ok1\n");
		if (triangAll == 0) {
			updateRhsG(hList, sMulList, sMulListComp, sMulListDest,
					   rhsLocal, vmListLocal,
					   h.nChannels, sChannelInfo,
					   sGateInfo, sGateState,
					   h.compListSize, sActiveCompList, h.eLeak,
					freeMem); // RYC
			backSubstituteG(hList, sLeftList, sLeftListLine, sLeftListColumn, rhsLocal, vmListLocal, freeMem); // RYC
		}
		else {

			upperTriangularizeAll(hList, sTriangList, sLeftList, sLeftListLine, sLeftListColumn,
					sLeftStartPos, rhsLocal, vmListLocal,
					   h.nChannels, sChannelInfo,
					   sGateInfo, sGateState,
					   h.compListSize, sActiveCompList, h.eLeak,
					   freeMem);
			backSubstituteG(hList, sTriangList, sLeftListLine, sLeftListColumn, rhsLocal, vmListLocal, freeMem); // RYC
		}
		//printf ("SolveMatrixG: Ok2\n");

		for (int k=0; k<nComp; k++) {
			int index = k * nSteps + gStep;
			h.vmTimeSerie[index] = vmListLocal[POS(k)]; // RYC
		}

		currStep = currStep + 1;


		if (vmListLocal[POS(nComp-1)] >= h.threshold && ((currStep * dt) - h.lastSpike) > h.minSpikeInterval) {

			h.spikeTimes[nGeneratedSpikes] = currStep * dt;
			h.lastSpike = currStep * dt;
			nGeneratedSpikes++;
		}

		h.currStep = currStep;
	}

	h.nGeneratedSpikes[neuron] = nGeneratedSpikes;

	//__syncthreads();

	for (int k=0; k<nComp; k++) {
		h.rhsM[k] = rhsLocal[POS(k)];
		h.vmList[k] = vmListLocal[POS(k)]; // RYC
	}

	for (int k=0; k < nGatesTotal; k ++ )
		h.gateState[k] = sGateState[POS(k)];

//	for (int k=0; k < h.compListSize; k++ ) {
//		h.n[k] = nGate[POS(k)];
//		h.h[k] = hGate[POS(k)];
//		h.m[k] = mGate[POS(k)];
//	}

	vmListGlobal[neuron] = vmListLocal[POS(nComp-1)];

	// used only for debugging
	//h.active[0] = ((char *)freeMem - (char *)sharedMem)*1.0;

}
