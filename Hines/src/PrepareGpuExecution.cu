#include "hip/hip_runtime.h"
#include "HinesMatrix.hpp"
#include "PlatformFunctions.hpp"
#include "HinesStruct.hpp"
#include "Connections.hpp"
#include "SpikeStatistics.hpp"
#include "GpuSimulationController.hpp"
#include <cassert>
#include <cstdlib>
#include <cstdio>
#include <cstring>
#include <pthread.h>

#ifdef MPI_GPU_NN
#include <mpi.h>
#endif

#include <hip/hip_runtime.h> // Necessary to allow better eclipse integration
#include <hip/hip_runtime_api.h> // Necessary to allow better eclipse integration
#include <> // Necessary to allow better eclipse integration
#include <hip/device_functions.h> // Necessary to allow better eclipse integration

extern __global__ void solveMatrixG(HinesStruct *hList, int nSteps, int nNeurons, ftype *spikeListGlobal, ftype *weightListGlobal, int *spikeListPosGlobal, int *spikeListStartGlobal, ftype *vmListGlobal);
extern ConnGpu* createGpuConnections( MPIConnectionInfo *connInfoList, int destType, int *nNeurons, int nGroups );
extern int **countReceivedSpikesCpu(ConnGpu *connGpuList, int nNeurons, int nGroups, ucomp **nGeneratedSpikes);
extern __global__ void performCommunicationsG(int nNeurons, ConnGpu *connGpuListDev, ucomp **nGeneratedSpikesDev, ftype **genSpikeTimeListDev,
		HinesStruct *hList, ftype *spikeListGlobal, ftype *weightListGlobal, int *spikeListPosGlobal, int *spikeListSizeGlobal,
		ftype *randomSpikeTimesDev, int *randomSpikeDestDev, int *nReceivedSpikesGlobal0, int *nReceivedSpikesGlobal1);
__global__ void performCommunicationsG_Step1(int nNeurons, ConnGpu *connGpuListDev, ucomp **nGeneratedSpikesDev, ftype **genSpikeTimeListDev,
		HinesStruct *hList, ftype *spikeListGlobal, ftype *weightListGlobal, int *spikeListPosGlobal, int *spikeListSizeGlobal,
		ftype *randomSpikeTimesDev, int *randomSpikeDestDev, ftype *tmpDevMemory);
__global__ void performCommunicationsG_Step2(int nNeurons, ConnGpu *connGpuListDev, ucomp **nGeneratedSpikesDev, ftype **genSpikeTimeListDev,
		HinesStruct *hList, ftype *spikeListGlobal, ftype *weightListGlobal, int *spikeListPosGlobal, int *spikeListSizeGlobal,
		ftype *randomSpikeTimesDev, int *randomSpikeDestDev, ftype *tmpDevMemory);


void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err)
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

//===================================================================================================

GpuSimulationController::GpuSimulationController(ThreadInfo *tInfo) {

	this->tInfo = tInfo;
	this->sharedData = tInfo->sharedData;
	this->kernelInfo = tInfo->sharedData->kernelInfo;
}

void GpuSimulationController::prepareSynapses() {

	int *nNeurons = tInfo->nNeurons;
	SynapticData* synData = sharedData->synData;

	/**
	 * Prepare the synaptic channels and spike generation
	 */
	int totalTypes = synData->totalTypes;

	pthread_mutex_lock (sharedData->mutex);
	if (synData->spikeListDevice == 0) {
		synData->spikeListDevice   = (ftype **) malloc (sizeof(ftype *) * totalTypes);
		synData->weightListDevice  = (ftype **) malloc (sizeof(ftype *) * totalTypes);
		synData->spikeListPosDevice  = (int **) malloc (sizeof(int *) * totalTypes);
		synData->spikeListSizeDevice = (int **) malloc (sizeof(int *) * totalTypes);


		synData->spikeListGlobal     = (ftype **) malloc (sizeof(ftype *) * totalTypes);
		synData->weightListGlobal    = (ftype **) malloc (sizeof(ftype *) * totalTypes);
		synData->spikeListPosGlobal  =  (int **) malloc (sizeof(int *) * totalTypes);
		synData->spikeListSizeGlobal = (int **) malloc (sizeof(int *) * totalTypes);

		synData->vmListHost    = (ftype **) malloc (sizeof(ftype *) * totalTypes);
		synData->vmListDevice  = (ftype **) malloc (sizeof(ftype *) * totalTypes);

		synData->genSpikeTimeListHost     = (ftype **) malloc (sizeof(ftype *)  * totalTypes);
		synData->genSpikeTimeListDevice   = (ftype **) malloc (sizeof(ftype *)  * totalTypes);

		synData->nGeneratedSpikesHost     = (ucomp **) malloc (sizeof(ucomp *)  * totalTypes);
		synData->nGeneratedSpikesDevice   = (ucomp **) malloc (sizeof(ucomp *)  * totalTypes);

		if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == GPU_COMM) {
			synData->genSpikeTimeListGpusDev  = (ftype ***) malloc (sizeof(ftype **) * sharedData->nThreadsCpu);
			synData->genSpikeTimeListGpusHost = (ftype ***) malloc (sizeof(ftype **) * sharedData->nThreadsCpu);
			synData->nGeneratedSpikesGpusDev  = (ucomp ***) malloc (sizeof(ucomp **) * sharedData->nThreadsCpu);
			synData->nGeneratedSpikesGpusHost = (ucomp ***) malloc (sizeof(ucomp **) * sharedData->nThreadsCpu);
		}
	}
	pthread_mutex_unlock (sharedData->mutex);


	/**
	 * Prepare the delivered spike related lists
	 * - spikeListPos and spikeListSize
	 */
	for (int type = tInfo->startTypeThread; type < tInfo->endTypeThread; type++) {
		synData->spikeListGlobal[type] = 0;
		synData->weightListGlobal[type] = 0;
		int totalListPosSize = 1 + sharedData->matrixList[type][0].synapticChannels->synapseListSize * nNeurons[type];
		synData->spikeListPosGlobal[type]  = (int *) malloc(sizeof(int) * totalListPosSize);
		synData->spikeListSizeGlobal[type] = (int *) malloc(sizeof(int) * (nNeurons[type]+1));
		hipMalloc ((void **) &(synData->spikeListPosDevice[type]),  sizeof(int)  * totalListPosSize);
		hipMalloc ((void **) &(synData->spikeListSizeDevice[type]), sizeof(int)  * (nNeurons[type]+1));

		synData->vmListHost[type] = (ftype *) malloc(sizeof(ftype) * nNeurons[type]);
		hipMalloc ((void **) &(synData->vmListDevice[type]), sizeof(ftype)  * nNeurons[type]);

		for (int i=0; i<totalListPosSize; i++)
			synData->spikeListPosGlobal[type][i] = 0;
		for (int i=0; i<nNeurons[type]+1; i++)
			synData->spikeListSizeGlobal[type][i] = 0;

		hipMemcpy(synData->spikeListPosDevice[type], synData->spikeListPosGlobal[type],
				sizeof(int) * totalListPosSize, hipMemcpyHostToDevice);
		hipMemcpy(synData->spikeListSizeDevice[type], synData->spikeListSizeGlobal[type],
				sizeof(int) * (nNeurons[type]+1), hipMemcpyHostToDevice);
	}

	/**
	 * Prepare the lists containing the generated spikes during each kernel call
	 */
	for (int type = tInfo->startTypeThread; type < tInfo->endTypeThread; type++) {
		int spikeTimeListSize = GENSPIKETIMELIST_SIZE;

		synData->genSpikeTimeListHost[type] = (ftype *) malloc(sizeof(ftype) * spikeTimeListSize * nNeurons[type]);
		hipMalloc ((void **) &(synData->genSpikeTimeListDevice[type]), sizeof(ftype) * spikeTimeListSize * nNeurons[type]);

		synData->nGeneratedSpikesHost[type] = (ucomp *) malloc(sizeof(ucomp) * nNeurons[type]);
		hipMalloc ((void **) &(synData->nGeneratedSpikesDevice[type]), sizeof(ucomp) * nNeurons[type]);

		for (int neuron = 0; neuron < nNeurons[type]; neuron++ ) {
			HinesStruct & h = sharedData->hList[type][neuron];
			h.spikeTimes  = synData->genSpikeTimeListDevice[type] + spikeTimeListSize * neuron;
			h.nGeneratedSpikes = synData->nGeneratedSpikesDevice[type];// + neuron;
		}
	}
}

int GpuSimulationController::prepareExecution(int type) {

	int nNeurons = tInfo->nNeurons[type];
	int nKernelSteps = kernelInfo->nKernelSteps;

	HinesStruct **hListPtr = &(sharedData->hList[type]);

	HinesStruct *hList = (HinesStruct *)malloc(nNeurons*sizeof(HinesStruct)); //new HinesStruct[nNeurons];

	HinesMatrix & m0 = sharedData->matrixList[type][0];
	int nComp = m0.nComp;
	int nCompActive = m0.activeChannels->getCompListSize();
	int nSynaptic = m0.synapticChannels->synapseListSize;

	int sharedMemMatrix    = sizeof(ftype) * (3*nComp + m0.mulListSize + m0.leftListSize); //+ nComp*nComp;
	int sharedMemSynaptic  = sizeof(ftype) * 4 * m0.synapticChannels->nChannelTypes;
	int sharedMemTotal = sharedMemMatrix + sharedMemSynaptic;

	int exclusiveMemMatrix   = sizeof(ftype) * (5*nComp + m0.leftListSize);
	int exclusiveMemActive   = sizeof(ftype) * (nCompActive*7);
	//int exclusiveMemSynaptic = sizeof(ftype) * m.spikeTimeListSize;
	int exclusiveMemTotal = exclusiveMemMatrix + exclusiveMemActive + sizeof(ftype)*nComp*nKernelSteps;

	/**
	 * Allocates memory to all neurons
	 */
	printf("TotalMem = %10.3f MB %d %d %d\n",(sharedMemTotal + exclusiveMemTotal * nNeurons)/(1024.*1024.), sharedMemTotal, exclusiveMemTotal, nNeurons);
	ftype *memory;
	hipMalloc((void **)(&(memory)), sharedMemTotal + exclusiveMemTotal * nNeurons);

	/**
	 * Copies the shared ftype data from matrix[0] to the GPU
	 */
	ftype *sharedMemMatrixAddress = m0.Cm;
	hipMemcpy(memory, sharedMemMatrixAddress, sharedMemMatrix, hipMemcpyHostToDevice);
	ftype *sharedMemSynapticAddress = m0.synapticChannels->tau;
	hipMemcpy(memory+sharedMemMatrix/sizeof(ftype), sharedMemSynapticAddress, sharedMemSynaptic, hipMemcpyHostToDevice);

	/**
	 * Allocates and copies shared data of type ucomp from matrix[0] to the GPU
	 */
	ucomp *ucompSharedMemory;
	hipMalloc((void **)(&(ucompSharedMemory)), sizeof(ucomp) * ((m0.mulListSize + m0.leftListSize) * 2 + nComp + nCompActive + 3*nSynaptic));
	ucomp *ucompActiveAddress   = ucompSharedMemory + (m0.mulListSize + m0.leftListSize) * 2 + nComp;
	ucomp *ucompSynapticAddress = ucompActiveAddress + nCompActive;

	hipMemcpy(ucompSharedMemory,    m0.ucompMemory, sizeof(ucomp) * ((m0.mulListSize + m0.leftListSize) * 2 + nComp), hipMemcpyHostToDevice);
	hipMemcpy(ucompActiveAddress,   m0.activeChannels->getCompList(), sizeof(ucomp) * nCompActive, hipMemcpyHostToDevice);
	hipMemcpy(ucompSynapticAddress, m0.synapticChannels->synapseCompList, sizeof(ucomp) * nSynaptic*3, hipMemcpyHostToDevice);

	/**
	 * Prepare the MatrixStruct h for each neuron in the GPU
	 */
	for (int neuron = 0; neuron < nNeurons; neuron++ ) {

		HinesMatrix & m = sharedData->matrixList[type][neuron];

		HinesStruct & h = hList[neuron];
		h.type = type;

		/**
		 * Shared memory
		 */
		h.memoryS = memory;
		h.Cm = h.memoryS;
		h.Ra = h.Cm + nComp;
		h.Rm = h.Ra + nComp;
		h.leftList = h.Rm + nComp;
		h.mulList  = h.leftList + m.leftListSize; // Used only when triangAll = 0

		/**
		 * Memory allocated per thread
		 */
		h.memoryE = memory + sharedMemTotal/sizeof(ftype) + neuron*exclusiveMemTotal/sizeof(ftype);
		ftype *exclusiveAddressM = m.rhsM;
		hipMemcpy(h.memoryE, exclusiveAddressM, exclusiveMemMatrix, hipMemcpyHostToDevice);
		// must match the order in HinesMatrix.cpp
		h.rhsM = h.memoryE	;
		h.vmList = h.rhsM + nComp;
		h.vmTmp = h.vmList + nComp;
		h.curr = h.vmTmp + nComp;
		h.active = h.curr + nComp;
		h.triangList = h.active + nComp; // triangularized list
		h.vmTimeSerie = h.triangList + m.leftListSize;

		h.currStep = m.currStep;
		h.vRest = m.vRest;
		h.dx = m.dx;
		h.nComp = m.nComp;
		h.dt = m.dt;
		h.triangAll = m.triangAll;

		h.mulListSize = m.mulListSize;
		h.leftListSize = m.leftListSize;
		h.mulListComp    = ucompSharedMemory;
		h.mulListDest    = h.mulListComp  + h.mulListSize;
		h.leftListLine   = h.mulListDest  + h.mulListSize;
		h.leftListColumn = h.leftListLine + h.leftListSize;
		h.leftStartPos   = h.leftListColumn + h.leftListSize;

		// must match the order in ActiveChannels.cpp
		if (nCompActive > 0) {
			hipMemcpy(h.vmTimeSerie + nComp*nKernelSteps, m.activeChannels->memory, exclusiveMemActive, hipMemcpyHostToDevice);

			h.n = h.vmTimeSerie + nComp*nKernelSteps;
			h.h = h.n + nCompActive;
			h.m = h.h + nCompActive;
			h.gNaBar = h.m + nCompActive;
			h.gKBar  = h.gNaBar + nCompActive;
			h.gNaChannel  = h.gKBar + nCompActive;
			h.gKChannel  = h.gNaChannel + nCompActive;

			h.ELeak = m.activeChannels->ELeak;
			h.EK = m.activeChannels->EK;
			h.ENa = m.activeChannels->ENa;

			h.compListSize = nCompActive;
			h.compList = ucompActiveAddress;

			checkCUDAError("Memory Allocation3:");
		}

		if (m.synapticChannels != 0) {

			h.synapseListSize = m.synapticChannels->synapseListSize;
			h.nChannelTypes   = m.synapticChannels->nChannelTypes;
			h.synapseCompList = ucompSynapticAddress;
			h.synapseTypeList = h.synapseCompList + h.synapseListSize;
			//h.synSpikeListPos = h.synapseTypeList + h.synapseListSize;

			h.spikeListSize 	= 0;
			h.spikeList     	= 0;
			h.synapseWeightList = 0;

			h.tau = memory+sharedMemMatrix/sizeof(ftype);
			h.gmax = h.tau  + 2 * m.synapticChannels->nChannelTypes;
			h.esyn = h.gmax + m.synapticChannels->nChannelTypes;

			h.lastSpike 		= m.lastSpike;
			h.spikeTimeListSize = m.spikeTimeListSize;
			h.threshold        = m.threshold;
			h.minSpikeInterval = m.minSpikeInterval;

		}
		h.nNeurons = nNeurons;
		sharedData->matrixList[type][neuron].freeMem();
	}

	*hListPtr = hList;

	return 0;
}

void GpuSimulationController::checkGpuCommunicationsSpikes(int spikeListSizeMax, int type) {

	int *nNeurons = tInfo->nNeurons;
	SynapticData *synData = sharedData->synData;
	int kernelSteps = kernelInfo->nKernelSteps;
	int synapseListSize = sharedData->matrixList[type][0].synapticChannels->synapseListSize;

    ftype *spkTmp = (ftype*)(malloc(sizeof (ftype) * spikeListSizeMax * nNeurons[type]));
    ftype *weightTmp = (ftype*)(malloc(sizeof (ftype) * spikeListSizeMax * nNeurons[type]));
    int *spikeListSizeTmp = (int*)(malloc(sizeof (int) * nNeurons[type]));
    int *spikeListPosTmp = (int*)(malloc(sizeof (int) * nNeurons[type] * synapseListSize));
    hipMemcpy(spkTmp, synData->spikeListDevice[type], sizeof (ftype) * spikeListSizeMax * nNeurons[type], hipMemcpyDeviceToHost);
    hipMemcpy(weightTmp, synData->weightListDevice[type], sizeof (ftype) * spikeListSizeMax * nNeurons[type], hipMemcpyDeviceToHost);
    hipMemcpy(spikeListSizeTmp, synData->spikeListSizeDevice[type], sizeof (int) * nNeurons[type], hipMemcpyDeviceToHost);
    hipMemcpy(spikeListPosTmp, synData->spikeListPosDevice[type], sizeof (int) * nNeurons[type] * synapseListSize, hipMemcpyDeviceToHost);
    for(int neuron = 0;neuron < nNeurons[type];neuron++){
        if(kStep > 100 && spikeListSizeTmp[neuron] != synData->spikeListSizeGlobal[type][neuron]){
            printf("SpikeListSIZE time=%f type=%d, neuron=%d, gpu=%d cpu=%d\n", sharedData->dt * (kStep + kernelSteps), type, neuron, spikeListSizeTmp[neuron], synData->spikeListSizeGlobal[type][neuron]);
            //assert (false);
        }
    }

    for(int neuronSyn = 0;neuronSyn < 2 * nNeurons[type];neuronSyn++){
        if(kStep > 100 && spikeListPosTmp[neuronSyn] != synData->spikeListPosGlobal[type][neuronSyn]){
            printf("SpikeListPOS time=%f type=%d, neuron=%d, syn=%d, gpu=%d cpu=%d\n", sharedData->dt * (kStep + kernelSteps), type, neuronSyn / 2, neuronSyn % 2, spikeListPosTmp[neuronSyn], synData->spikeListPosGlobal[type][neuronSyn]);
            //assert (false);
        }
    }

    for (int neuronSyn=0; neuronSyn<2*nNeurons[type]; neuronSyn++) {

					int localPos = sharedData->synData->spikeListPosGlobal[type][neuronSyn];

					int nSpikes  = 0;
					if (neuronSyn%2 == 0)
						nSpikes = sharedData->synData->spikeListPosGlobal[type][neuronSyn+1];
					else
						nSpikes = sharedData->synData->spikeListSizeGlobal[type][neuronSyn/2] -
						sharedData->synData->spikeListPosGlobal[type][neuronSyn];

					// threshold = 24ms

					for (int spk = 0; spk < nSpikes; spk++) {

						int globalPos = (localPos + spk) * nNeurons[type] + neuronSyn/2;

						int spk2 = 0;
						for (; spk2 < nSpikes; spk2++) {
							int globalPos2 = (localPos + spk2) * nNeurons[type] + neuronSyn/2;
							if ( (spkTmp[globalPos] == synData->spikeListGlobal[type][globalPos2]) &&
									(weightTmp[globalPos] == synData->weightListGlobal[type][globalPos2]) )
								break;
						}

						if ( spk2 == nSpikes ) {

							printf("time=%f type=%d, neuron=%d, syn=%d, spk=%d, nSpikes=%d, "
									"firstDelievered=%d, nRandom=%d, gpu=%f cpu=%f\n",
									sharedData->dt*(kStep+kernelSteps), type, neuronSyn/2, neuronSyn%2, spk, nSpikes,
									sharedData->matrixList[type][neuronSyn/2].synapticChannels->nDelieveredSpikes[neuronSyn%2],
									sharedData->matrixList[type][neuronSyn/2].synapticChannels->nRandom,
									spkTmp[globalPos], synData->spikeListGlobal[type][globalPos]);

							if (spk > 0) {
								printf("gpuPrevious=%f cpuPrevious=%f\n",
										spkTmp[globalPos-nNeurons[type]], synData->spikeListGlobal[type][globalPos-nNeurons[type]]);

							}
							printf("nNeuronsGroup0=%d, nThreadsConn=%d, nConnections=%d\n",
									sharedData->connGpuListHost[type][0].nNeuronsGroup, kernelInfo->nThreadsComm[type],
									sharedData->connGpuListHost[type][0].nConnectionsTotal);

							printf("GPU\n");
							for (int iSpk = 0; iSpk < nSpikes; iSpk++) {
								int globalPos = (localPos + iSpk) * nNeurons[type] + neuronSyn/2;
								printf("%7.3f ",spkTmp[globalPos]);
							}
							printf("\nCPU\n");
							for (int iSpk = 0; iSpk < nSpikes; iSpk++) {
								int globalPos = (localPos + iSpk) * nNeurons[type] + neuronSyn/2;
								printf("%7.3f ",synData->spikeListGlobal[type][globalPos]);
							}
							printf("\nGPU\n");
							for (int iSpk = 0; iSpk < nSpikes; iSpk++) {
								int globalPos = (localPos + iSpk) * nNeurons[type] + neuronSyn/2;
								printf("%7.3f ",weightTmp[globalPos]);
							}
							printf("\nCPU\n");
							for (int iSpk = 0; iSpk < nSpikes; iSpk++) {
								int globalPos = (localPos + iSpk) * nNeurons[type] + neuronSyn/2;
								printf("%7.3f ",synData->weightListGlobal[type][globalPos]);
							}
							printf("\n");

							//sharedData->connection->getConnArray(type );

							assert(false);
						}
					}
				}
    free(spkTmp);
    free(spikeListSizeTmp);
    free(spikeListPosTmp);
}

void GpuSimulationController::updateBenchmark()
{
    if(benchConf.gpuCommMode == CPU_COMM) {

				bench.totalHinesKernel	+= (bench.kernelFinish 	- bench.kernelStart)/1000.;
				bench.totalConnRead	  	+= (bench.connRead 		- bench.kernelFinish)/1000.;
				bench.totalConnWait		+= (bench.connWait 		- bench.connRead)/1000.;
				bench.totalConnWrite	+= (bench.connWrite 	- bench.connWait)/1000.;
			}
			else if (benchConf.gpuCommBenchMode == GPU_COMM_SIMPLE) {
				bench.totalHinesKernel	+= (bench.kernelFinish 	- bench.kernelStart)/1000.;
				bench.totalConnWait		+= (bench.connWait 		- bench.kernelFinish)/1000.;
				bench.totalConnRead	  	+=  bench.connRead  / 1000.;
				bench.totalConnWrite	+= (bench.connWrite - bench.connWait - bench.connRead)/1000.;
			}
			else if (benchConf.gpuCommBenchMode == GPU_COMM_DETAILED) {
				bench.totalHinesKernel	+= (bench.kernelFinish 	- bench.kernelStart)/1000.;
				bench.totalConnWait		+= (bench.connWait 		- bench.kernelFinish)/1000.;
				bench.totalConnRead	  	+=  bench.connRead  / 1000.;
				bench.totalConnWrite	+=  bench.connWrite / 1000.;
			}
}

int GpuSimulationController::updateSpikeListSizeGlobal(int type, int maxSpikesNeuron)
{

	int spikeListSizeMax = 0;
	for (int n = 0; n < tInfo->nNeurons[type]; n++) {

		int synapsePosListTmp = 0;
		int synapseListSize = sharedData->matrixList[type][0].synapticChannels->synapseListSize;
		HinesMatrix & m = sharedData->matrixList[type][n];

		if (m.synapticChannels->spikeListSize > spikeListSizeMax)
			spikeListSizeMax = m.synapticChannels->spikeListSize;

		/**
		 * Stores the number of spikes delivered to each neuron
		 */
		sharedData->synData->spikeListSizeGlobal[type][n] = m.synapticChannels->spikeListSize;

		/**
		 * Copies the information about the start position of the spikes at each synapse
		 */
		for (int i=0; i<synapseListSize; i++)
			sharedData->synData->spikeListPosGlobal[type][synapsePosListTmp + i] =
					m.synapticChannels->synSpikeListPos[i];
		synapsePosListTmp += synapseListSize;


		if (spikeListSizeMax > maxSpikesNeuron) {
			printf ("Neuron with %d spikes, more than the max of %d.\n", spikeListSizeMax, maxSpikesNeuron);
			assert(false);
		}
	}

	return spikeListSizeMax;
}

void GpuSimulationController::transferSynapticSpikeInfoToGpu(int type, int spikeListSizeMax) {

	SynapticData *& synData = sharedData->synData;
    int synapseListSize = sharedData->matrixList[type][0].synapticChannels->synapseListSize;

    checkCUDAError("cp2a:");
    hipMemcpy(synData->spikeListDevice[type], synData->spikeListGlobal[type], sizeof (ftype) * spikeListSizeMax * tInfo->nNeurons[type], hipMemcpyHostToDevice);
    checkCUDAError("cp2b:");
    hipMemcpy(synData->weightListDevice[type], synData->weightListGlobal[type], sizeof (ftype) * spikeListSizeMax * tInfo->nNeurons[type], hipMemcpyHostToDevice);
    checkCUDAError("cp2c:");
    hipMemcpy(synData->spikeListPosDevice[type], synData->spikeListPosGlobal[type], sizeof (int) * (1 + synapseListSize * tInfo->nNeurons[type]), hipMemcpyHostToDevice);
    checkCUDAError("cp2d:");
    hipMemcpy(synData->spikeListSizeDevice[type], synData->spikeListSizeGlobal[type], sizeof (int) * (tInfo->nNeurons[type] + 1), hipMemcpyHostToDevice);
    checkCUDAError("cp2e:");
}

void GpuSimulationController::mpiAllGatherConnections()
{
    // Sends the number of connections from each process
    int *countConnProc = new int[tInfo->nProcesses];
    MPI_Allgather( &(sharedData->connInfo->nConnections), 1, MPI_INT, countConnProc, 1, MPI_INT, MPI_COMM_WORLD);
    // Just for testing
    printf("Connections: %d/%d [", tInfo->currProcess, sharedData->connInfo->nConnections);
    for(int p = 0;p < tInfo->nProcesses;p++)
        printf(" %d |", countConnProc[p]);

    printf("]\n");
    // Allocates the buffer to put information about all the connections
    int countTotalAllProc = 0;
    int *displaceConnProc = new int[tInfo->nProcesses];
    displaceConnProc[0] = 0;
    for(int p = 0;p < tInfo->nProcesses;p++){
        countTotalAllProc += countConnProc[p];
        if(p > 0)
            displaceConnProc[p] = displaceConnProc[p - 1] + countConnProc[p - 1];

    }
    MPIConnectionInfo *connInfoAllProc = new MPIConnectionInfo;
    connInfoAllProc->nConnections = countTotalAllProc;
    connInfoAllProc->source = new int[countTotalAllProc];
    connInfoAllProc->dest = new int[countTotalAllProc];
    connInfoAllProc->synapse = new ucomp[countTotalAllProc];
    connInfoAllProc->weigth = new ftype[countTotalAllProc];
    connInfoAllProc->delay = new ftype[countTotalAllProc];
    // Sends the data for each part of the connection list
    // int *source; int *dest; ucomp *synapse; ftype *weigth; ftype *delay;
    MPI_Allgatherv( sharedData->connInfo->source, sharedData->connInfo->nConnections, MPI_INT,
				connInfoAllProc->source, countConnProc, displaceConnProc, MPI_INT, MPI_COMM_WORLD);
    MPI_Allgatherv( sharedData->connInfo->dest, sharedData->connInfo->nConnections, MPI_INT,
				connInfoAllProc->dest, countConnProc, displaceConnProc, MPI_INT, MPI_COMM_WORLD);
    MPI_Allgatherv( sharedData->connInfo->synapse, sharedData->connInfo->nConnections, MPI_UCOMP,
				connInfoAllProc->synapse, countConnProc, displaceConnProc, MPI_UCOMP, MPI_COMM_WORLD);
    MPI_Allgatherv( sharedData->connInfo->weigth, sharedData->connInfo->nConnections, MPI_FTYPE,
				connInfoAllProc->weigth, countConnProc, displaceConnProc, MPI_FTYPE, MPI_COMM_WORLD);
    MPI_Allgatherv( sharedData->connInfo->delay, sharedData->connInfo->nConnections, MPI_FTYPE,
				connInfoAllProc->delay, countConnProc, displaceConnProc, MPI_FTYPE, MPI_COMM_WORLD);
    sharedData->connection->clearMPIConnections(sharedData->connInfo);
    sharedData->connInfo = connInfoAllProc;
}

void GpuSimulationController::syncCpuThreads()
{
    pthread_mutex_lock(sharedData->mutex);
    sharedData->nBarrier++;
    if(sharedData->nBarrier < sharedData->nThreadsCpu)
        pthread_cond_wait(sharedData->cond, sharedData->mutex);

    else{
        sharedData->nBarrier = 0;
        pthread_cond_broadcast(sharedData->cond);
    }
    pthread_mutex_unlock(sharedData->mutex);
}

void GpuSimulationController::generateRandomSpikes( int type, RandomSpikeInfo & randomSpkInfo )
{

	int kernelSteps = kernelInfo->nKernelSteps;
	ftype dt = sharedData->dt;

	randomSpkInfo.nRandom = 0;
    for (int neuron = 0; neuron < tInfo->nNeurons[type]; neuron++) {
				HinesMatrix & m = sharedData->matrixList[type][neuron];

				if ((kStep + kernelSteps)*m.dt > 9.9999 && sharedData->typeList[type] == PYRAMIDAL_CELL) {
					int32_t spkTime;

					random_r(sharedData->randBuf[tInfo->threadNumber], &spkTime);
					ftype rate = (sharedData->inputSpikeRate) * (kernelSteps * dt);
					if (spkTime / (float) RAND_MAX < rate ) {
						spkTime = (kStep + kernelSteps)*dt + (ftype)spkTime/RAND_MAX * (kernelSteps * dt);

						if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == GPU_COMM) {
							assert(randomSpkInfo.nRandom < randomSpkInfo.listSize);
							randomSpkInfo.spikeTimes[randomSpkInfo.nRandom] = spkTime;
							randomSpkInfo.spikeDest[randomSpkInfo.nRandom] = neuron;
						}
						randomSpkInfo.nRandom++;
						if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == CPU_COMM)
							m.synapticChannels->addSpike(0, spkTime, 1);
					}
				}
			}
}

void GpuSimulationController::performCPUCommunication(int type, int maxSpikesNeuron, int nRandom) {

    int totalNumberSpikes = 0;
    for(int neuron = 0;neuron < tInfo->nNeurons[type];neuron++){
        HinesMatrix & m = sharedData->matrixList[type][neuron];
        /**
         * Updates the spike list when using CPU communications
         */
        m.synapticChannels->updateSpikeListGpu(sharedData->dt * (kStep + kernelInfo->nKernelSteps),
        		sharedData->synData->spikeListGlobal[type], sharedData->synData->weightListGlobal[type],
        		maxSpikesNeuron, tInfo->nNeurons[type], neuron, type);

        totalNumberSpikes += m.synapticChannels->spikeListSize;
        // Used to print spike statistics in the end of the simulation
        sharedData->spkStat->addReceivedSpikes(type, neuron, m.synapticChannels->getAndResetNumberOfAddedSpikes() - nRandom);
    }
    // The max number of spikes delivered to a single neuron
    int spikeListSizeMax = updateSpikeListSizeGlobal(type, maxSpikesNeuron);

    /**
     * Tests if the list of spikes, positions and list sizes are the same
     * for the GPU and CPU communications
     */
    if (benchConf.gpuCommMode == CHK_COMM )
					checkGpuCommunicationsSpikes(spikeListSizeMax, type);
    /**
				 * Copy synaptic and spike info into the GPU
				 */
    if (benchConf.gpuCommMode == CPU_COMM)
					transferSynapticSpikeInfoToGpu(type, spikeListSizeMax);
}

void GpuSimulationController::performGPUCommunications(int type, RandomSpikeInfo & randomSpkInfo) {

	/**
	 * Transfers the list of random spikes to the GPU
	 */
	for(int i = randomSpkInfo.nRandom;i < randomSpkInfo.listSize;i++){
		randomSpkInfo.spikeTimes[i] = -1;
		randomSpkInfo.spikeDest[i] = -1;
	}

	SynapticData *synData = sharedData->synData;
	int threadNumber = tInfo->threadNumber;
	int *nBlocksComm  = kernelInfo->nBlocksComm;
	int *nThreadsComm = kernelInfo->nThreadsComm;

	ftype *randomSpikeTimesDev;
	int *randomSpikeDestDev;
	hipMalloc((void**)(&randomSpikeTimesDev), sizeof (ftype) * randomSpkInfo.listSize);
	hipMalloc((void**)(&randomSpikeDestDev), sizeof (int) * randomSpkInfo.listSize);
	hipMemcpy(randomSpikeTimesDev, randomSpkInfo.spikeTimes, sizeof (ftype) * randomSpkInfo.listSize, hipMemcpyHostToDevice);
	hipMemcpy(randomSpikeDestDev, randomSpkInfo.spikeDest, sizeof (int) * randomSpkInfo.listSize, hipMemcpyHostToDevice);

	if (benchConf.gpuCommBenchMode == GPU_COMM_SIMPLE) {

		uint64 connTmp = 0;
		if (threadNumber == 0 && benchConf.gpuCommBenchMode == GPU_COMM_SIMPLE)
			connTmp = gettimeInMilli();

		/**
		 * TODO: Remove Me [MPI]
		 * Used only during debugging to check the number of received spikes per process
		 */
		int *nReceivedSpikesDev0, *nReceivedSpikesDev1, *nReceivedSpikesHost0, *nReceivedSpikesHost1;
		hipMalloc ( (void **) &nReceivedSpikesDev0, sizeof(int) * nBlocksComm[type]);
		hipMalloc ( (void **) &nReceivedSpikesDev1, sizeof(int) * nBlocksComm[type]);
		nReceivedSpikesHost0 = (int *)malloc(sizeof(int) * nBlocksComm[type]);
		nReceivedSpikesHost1 = (int *)malloc(sizeof(int) * nBlocksComm[type]);

		//printf (" type=%d %d %d\n", nNeurons[type], nThreadsComm, sharedMemSizeComm);
		performCommunicationsG <<<nBlocksComm[type], nThreadsComm[type], kernelInfo->sharedMemSizeComm>>>(
				tInfo->nNeurons[type], sharedData->connGpuListDevice[type],
				synData->nGeneratedSpikesGpusDev[threadNumber], synData->genSpikeTimeListGpusDev[threadNumber],
				sharedData->hGpu[type], synData->spikeListDevice[type], synData->weightListDevice[type],
				synData->spikeListPosDevice[type], synData->spikeListSizeDevice[type],
				randomSpikeTimesDev, randomSpikeDestDev, nReceivedSpikesDev0, nReceivedSpikesDev1);

		/**
		 * TODO: Remove Me [MPI]
		 * Used only during debugging to check the number of received spikes per process
		 */
		hipMemcpy(nReceivedSpikesHost0, nReceivedSpikesDev0, sizeof(int) * nBlocksComm[type], hipMemcpyDeviceToHost);
		hipMemcpy(nReceivedSpikesHost1, nReceivedSpikesDev1, sizeof(int) * nBlocksComm[type], hipMemcpyDeviceToHost);
		long nReceived0 = 0, nReceived1 = 0;
		for (int k=0; k < nBlocksComm[type]; k++ ) {
			nReceived0 += nReceivedSpikesHost0[k];
			nReceived1 += nReceivedSpikesHost1[k];
		}
		printf("nReceived=0:%5ld|1:%5ld for type %d\n", nReceived0, nReceived1, type);
		hipFree ( nReceivedSpikesDev0);
		hipFree ( nReceivedSpikesDev1);
		free(nReceivedSpikesHost0);
		free(nReceivedSpikesHost1);


		if (threadNumber == 0 && benchConf.gpuCommBenchMode == GPU_COMM_SIMPLE) {
			hipDeviceSynchronize();
			bench.connRead += (gettimeInMilli()-connTmp);
		}

	}

	else if (benchConf.gpuCommBenchMode == GPU_COMM_DETAILED) {

		ftype *tmpDevMemory;
		hipMalloc ( (void **) &tmpDevMemory, sizeof(ftype) * tInfo->nNeurons[type] * 2 );

		uint64 connTmp = 0;
		if (threadNumber == 0)
			connTmp = gettimeInMilli();
		//int sharedMemComSize = sharedMemorySize/10;
		performCommunicationsG_Step1 <<<nBlocksComm[type], nThreadsComm[type], kernelInfo->sharedMemSizeComm>>> (
				tInfo->nNeurons[type], sharedData->connGpuListDevice[type],
				synData->nGeneratedSpikesGpusDev[threadNumber], synData->genSpikeTimeListGpusDev[threadNumber],
				sharedData->hGpu[type], synData->spikeListDevice[type], synData->weightListDevice[type],
				synData->spikeListPosDevice[type], synData->spikeListSizeDevice[type],
				randomSpikeTimesDev, randomSpikeDestDev, tmpDevMemory);

		hipDeviceSynchronize();

		if (threadNumber == 0) {
			bench.connRead += (gettimeInMilli()-connTmp);
			connTmp = gettimeInMilli();
		}

		performCommunicationsG_Step2 <<<nBlocksComm[type], nThreadsComm[type], kernelInfo->sharedMemSizeComm>>> (
				tInfo->nNeurons[type], sharedData->connGpuListDevice[type],
				synData->nGeneratedSpikesGpusDev[threadNumber], synData->genSpikeTimeListGpusDev[threadNumber],
				sharedData->hGpu[type], synData->spikeListDevice[type], synData->weightListDevice[type],
				synData->spikeListPosDevice[type], synData->spikeListSizeDevice[type],
				randomSpikeTimesDev, randomSpikeDestDev, tmpDevMemory);

		hipDeviceSynchronize();

		if (threadNumber == 0)
			bench.connWrite += (gettimeInMilli()-connTmp);

		hipFree (tmpDevMemory);
	}
	//int **countReceivedSpikesCpu(synData->v, nNeurons[type], nBlocksComm[type], synData->nGeneratedSpikesHost);
	hipDeviceSynchronize();
	hipFree(randomSpikeTimesDev);
	hipFree(randomSpikeDestDev);
	hipDeviceSynchronize();
}

void GpuSimulationController::checkVmValues()
{
	SynapticData *synData = sharedData->synData;
    for (int type = tInfo->startTypeThread; type < tInfo->endTypeThread; type++)
					for (int n = 0; n < tInfo->nNeurons[type]; n++)
						if ( synData->vmListHost[type][n] < -500 || 500 < synData->vmListHost[type][n] || synData->vmListHost[type][n] == 0.000000000000) {
							printf("********* type=%d neuron=%d %.2f\n", type, n, synData->vmListHost[type][n]);
							assert(false);
						}
}



void GpuSimulationController::prepareGpuSpikeDeliveryStructures()
{
	SynapticData *synData = sharedData->synData;
	int threadNumber = tInfo->threadNumber;

    synData->genSpikeTimeListGpusHost[threadNumber] = (ftype**)(malloc(sizeof (ftype*) * tInfo->totalTypes));
    for (int type = 0; type < tInfo->totalTypes; type++) {
			int genSpikeTimeListSize = GENSPIKETIMELIST_SIZE;
			// The device memory for the types of the current thread are already allocated
			if (tInfo->startTypeThread <= type && type < tInfo->endTypeThread)
				synData->genSpikeTimeListGpusHost[threadNumber][type] = synData->genSpikeTimeListDevice[type];
			else
				hipMalloc ((void **) &(synData->genSpikeTimeListGpusHost[threadNumber][type]),
						sizeof(ftype) * genSpikeTimeListSize * tInfo->nNeurons[type]);
		}
    // Copies the list of pointers to the genSpikeLists of each type
    hipMalloc((void**)(&synData->genSpikeTimeListGpusDev[threadNumber]), sizeof (ftype*) * tInfo->totalTypes);
    hipMemcpy(synData->genSpikeTimeListGpusDev[threadNumber], synData->genSpikeTimeListGpusHost[threadNumber], sizeof (ftype*) * tInfo->totalTypes, hipMemcpyHostToDevice);
    synData->nGeneratedSpikesGpusHost[threadNumber] = (ucomp**)(malloc(sizeof (ucomp*) * tInfo->totalTypes));
    for(int type = 0;type < tInfo->totalTypes;type++){
        if(tInfo->startTypeThread <= type && type < tInfo->endTypeThread)
            synData->nGeneratedSpikesGpusHost[threadNumber][type] = synData->nGeneratedSpikesDevice[type];

        else
            hipMalloc((void**)(&(synData->nGeneratedSpikesGpusHost[threadNumber][type])), sizeof (ucomp) * tInfo->nNeurons[type]);

    }
    // Copies the list of pointers to the nGeneratedSpikes of each type
    hipMalloc((void**)(&synData->nGeneratedSpikesGpusDev[threadNumber]), sizeof (ucomp*) * tInfo->totalTypes);
    hipMemcpy(synData->nGeneratedSpikesGpusDev[threadNumber], synData->nGeneratedSpikesGpusHost[threadNumber], sizeof (ucomp*) * tInfo->totalTypes, hipMemcpyHostToDevice);
}

void GpuSimulationController::createGpuCommunicationStructures()
{

	int *nBlocksComm = kernelInfo->nBlocksComm;

    for(int destType = tInfo->startTypeThread;destType < tInfo->endTypeThread;destType++){
        sharedData->connGpuListHost[destType] =
        		createGpuConnections(sharedData->connInfo, destType, tInfo->nNeurons, nBlocksComm[destType]);

        hipMalloc((void**)(&(sharedData->connGpuListDevice[destType])), nBlocksComm[destType] * sizeof (ConnGpu));
        hipMemcpy(sharedData->connGpuListDevice[destType], sharedData->connGpuListHost[destType], nBlocksComm[destType] * sizeof (ConnGpu), hipMemcpyHostToDevice);

        kernelInfo->nThreadsComm[destType] = sharedData->connGpuListHost[destType][0].nNeuronsGroup;
    }
}

void GpuSimulationController::addReceivedSpikesToTargetChannelCPU()
{

	hipDeviceSynchronize(); // Used to prevent reads of host memory before the copy

	for (int type = tInfo->startTypeThread; type < tInfo->endTypeThread; type++) {

		HinesStruct & hType = sharedData->hList[type][0];

		for (int source = 0; source < tInfo->nNeurons[type]; source++) {

			ucomp nGeneratedSpikes = sharedData->synData->nGeneratedSpikesHost[type][source];
			if (nGeneratedSpikes > 0) {

				ftype *spikeTimes = sharedData->synData->genSpikeTimeListHost[type] + hType.spikeTimeListSize * source;

				// Used to print spike statistics in the end of the simulation
				sharedData->spkStat->addGeneratedSpikes(type, source, spikeTimes, nGeneratedSpikes);

				std::vector<Conn> & connList = sharedData->connection->getConnArray(source + type*CONN_NEURON_TYPE);
				for (int conn=0; conn<connList.size(); conn++) {
					Conn & connStruct = connList[conn];
					SynapticChannels *targetSynapse = sharedData->matrixList[ connStruct.dest / CONN_NEURON_TYPE ][ connStruct.dest % CONN_NEURON_TYPE ].synapticChannels;
					targetSynapse->addSpikeList(connStruct.synapse, nGeneratedSpikes, spikeTimes, connStruct.delay, connStruct.weigth);
				}

			}
		}
	}
}

void GpuSimulationController::copyGeneratedSpikeListsToGPU()
{
	if(benchConf.verbose == 1)
		printf("Updating SpikeList %d\n", tInfo->threadNumber);

	for (int type=0; type < tInfo->totalTypes; type++) {
		if (type < tInfo->startTypeThread || tInfo->endTypeThread <= type) {

			hipMemcpy(sharedData->synData->genSpikeTimeListGpusHost[tInfo->threadNumber][type],
					sharedData->synData->genSpikeTimeListHost[type],
					sizeof(ftype) * GENSPIKETIMELIST_SIZE *tInfo-> nNeurons[type], hipMemcpyHostToDevice);

			hipMemcpy(sharedData->synData->nGeneratedSpikesGpusHost[tInfo->threadNumber][type],
					sharedData->synData->nGeneratedSpikesHost[type],
					sizeof(ucomp) * tInfo->nNeurons[type], hipMemcpyHostToDevice);
		}
	}
}

void GpuSimulationController::readGeneratedSpikesFromGPU()
{
    /*--------------------------------------------------------------
		 * Reads information from spike sources
		 *--------------------------------------------------------------*/
    if(benchConf.verbose == 1)
        printf("Getting spikes %d\n", tInfo->threadNumber);

    for(int type = tInfo->startTypeThread;type < tInfo->endTypeThread;type++){
        hipMemcpy(sharedData->synData->genSpikeTimeListHost[type], sharedData->synData->genSpikeTimeListDevice[type],
        		sizeof (ftype) * sharedData->hList[type][0].spikeTimeListSize * tInfo->nNeurons[type], hipMemcpyDeviceToHost);

        hipMemcpy(sharedData->synData->nGeneratedSpikesHost[type], sharedData->synData->nGeneratedSpikesDevice[type],
        		sizeof (ucomp) * tInfo->nNeurons[type], hipMemcpyDeviceToHost);

        checkCUDAError("Synapses2:");
    }
}

void GpuSimulationController::broadcastGeneratedSpikesMPISync()
{
    /*--------------------------------------------------------------
		 * [MPI] Send the list of generated spikes to other processes
		 *--------------------------------------------------------------*/
    if (tInfo->threadNumber == 0) {
			for (int type=0; type < tInfo->totalTypes; type++) {
				int genSpikeListTypeSize = GENSPIKETIMELIST_SIZE * tInfo->nNeurons[type];
				MPI_Bcast (sharedData->synData->genSpikeTimeListHost[type], genSpikeListTypeSize, MPI_FTYPE, tInfo->typeProcess[type], MPI_COMM_WORLD);
				MPI_Bcast (sharedData->synData->nGeneratedSpikesHost[type], tInfo->nNeurons[type],       MPI_UCOMP, tInfo->typeProcess[type], MPI_COMM_WORLD);
			}
		}
    // Synchronizes the thread to wait for the communication
    syncCpuThreads();
}

void GpuSimulationController::defineThreadTypes()
{
	int nThreadsCpu 	= sharedData->nThreadsCpu;
    int nTypesPerThread = (tInfo->totalTypes / (nThreadsCpu * tInfo->nProcesses));
    tInfo->startTypeThread = (tInfo->threadNumber + (tInfo->currProcess * nThreadsCpu)) * nTypesPerThread;
    tInfo->endTypeThread = (tInfo->threadNumber + 1 + (tInfo->currProcess * nThreadsCpu)) * nTypesPerThread;
    tInfo->startTypeProcess = tInfo->currProcess * nThreadsCpu * nTypesPerThread;
    tInfo->endTypeProcess = (tInfo->currProcess + 1) * nThreadsCpu * nTypesPerThread;

    int typeProcessCurr = 0;
    tInfo->typeProcess = new int[tInfo->totalTypes];
    for(int type = 0;type < tInfo->totalTypes;type++){
        if(type / ((typeProcessCurr + 1) * nThreadsCpu * nTypesPerThread) == 1)
            typeProcessCurr++;

        tInfo->typeProcess[type] = typeProcessCurr;
    }
}

void GpuSimulationController::createNeurons()
{
    /**------------------------------------------------------------------------------------
	 * Creates the neurons that will be simulated by the threads
	 *-------------------------------------------------------------------------------------*/
    for(int type = tInfo->startTypeThread;type < tInfo->endTypeThread;type++){
        int nComp = tInfo->nComp[type];
        int nNeurons = tInfo->nNeurons[type];
        //printf("process = %d | threadNumber = %d | type = %d nComp=%d nNeurons=%d seed=%d\n", tInfo->currProcess, tInfo->threadNumber, type, nComp, nNeurons, tInfo->sharedData->globalSeed);
        sharedData->matrixList[type] = new HinesMatrix[nNeurons];
        //HinesMatrix *mList = mListPtr[0];
        for(int n = 0;n < nNeurons;n++){
            HinesMatrix & m = sharedData->matrixList[type][n];
            if(nComp == 1)
                m.defineNeuronCableSquid();

            else
                m.defineNeuronTreeN(nComp, 1);

            m.createTestMatrix();
        }
    }

}

void GpuSimulationController::configureGpuKernel()
{

    /*--------------------------------------------------------------
	 * Select the device attributed to each thread
	 *--------------------------------------------------------------*/
    int nDevices = 0;
    hipGetDeviceCount(&nDevices);
    hipSetDevice((tInfo->threadNumber + (2 * tInfo->currProcess) + 2) % nDevices);
    hipGetDevice(&(tInfo->deviceNumber));
    tInfo->prop = new struct hipDeviceProp_t;
    hipGetDeviceProperties(tInfo->prop, tInfo->deviceNumber);

    checkCUDAError("Device selection:");
    //--------------------------------------------------------------
    /*--------------------------------------------------------------
	 * Configure number of threads and shared memory size for each kernel
	 *--------------------------------------------------------------*/
    kernelInfo->maxThreadsProc = 64;
    kernelInfo->maxThreadsComm = 16;
    kernelInfo->sharedMemSizeProc = 15 * 1024; // Valid for capability 1.x (16kB)
    kernelInfo->sharedMemSizeComm = 15 * 1024; // Valid for capability 1.x (16kB)
    if(tInfo->prop->major == 2){
        kernelInfo->maxThreadsProc = 256;
        kernelInfo->maxThreadsComm = 32; // or can be 64
        kernelInfo->sharedMemSizeProc = 47 * 1024; // Valid for capability 2.x (48kB)
        kernelInfo->sharedMemSizeComm = 15 * 1024; // Valid for capability 2.x (48kB)
    }
    //--------------------------------------------------------------
    for(int type = tInfo->startTypeThread;type < tInfo->endTypeThread;type++){
        // Number of blocks: multiple of #GPU multiprocessors and respects maxThreadsProc condition
        kernelInfo->nBlocksProc[type] = tInfo->prop->multiProcessorCount * (tInfo->nNeurons[type] / kernelInfo->maxThreadsProc / tInfo->prop->multiProcessorCount);
        if(tInfo->nNeurons[type] % kernelInfo->maxThreadsProc != 0 || (tInfo->nNeurons[type] / kernelInfo->maxThreadsProc) % kernelInfo->maxThreadsProc != 0)
            kernelInfo->nBlocksProc[type] += tInfo->prop->multiProcessorCount;

    }

    for(int destType = tInfo->startTypeThread;destType < tInfo->endTypeThread;destType++){

    	// Number of blocks: multiple of #GPU multiprocessors and respects maxThreadsComm condition
    	kernelInfo->nBlocksComm[destType] = tInfo->prop->multiProcessorCount * (tInfo->nNeurons[destType] / kernelInfo->maxThreadsComm / tInfo->prop->multiProcessorCount);
        if(tInfo->nNeurons[destType] % kernelInfo->maxThreadsComm != 0 || (tInfo->nNeurons[destType] / kernelInfo->maxThreadsComm) % kernelInfo->maxThreadsComm != 0)
        	kernelInfo->nBlocksComm[destType] += tInfo->prop->multiProcessorCount;
    }

    //	if (nComp0 <= 4) nThreads = (sizeof (ftype) == 4) ? 196 : 96;
    //	else if (nComp0 <= 8) nThreads = (sizeof (ftype) == 4) ? 128 : 64;
    //	else if (nComp0 <= 12) nThreads = (sizeof (ftype) == 4) ? 96 : 32;
    //	else if (nComp0 <= 16) nThreads = (sizeof (ftype) == 4) ? 64 : 32;

}

void GpuSimulationController::updateSharedDataInfo()
{
    sharedData->dt = sharedData->matrixList[tInfo->startTypeThread][0].dt;
    sharedData->hList = new HinesStruct*[tInfo->totalTypes];
    sharedData->hGpu = new HinesStruct*[tInfo->totalTypes];
    sharedData->synData = new SynapticData;
    sharedData->synData->spikeListDevice = 0;
    sharedData->synData->totalTypes = tInfo->totalTypes;
    sharedData->neuronInfoWriter = new NeuronInfoWriter(tInfo);
    sharedData->spkStat = new SpikeStatistics(tInfo->nNeurons, tInfo->totalTypes, tInfo->sharedData->typeList, tInfo->startTypeProcess, tInfo->endTypeProcess);
    kernelInfo->nThreadsComm = new int[tInfo->totalTypes];
    kernelInfo->nBlocksComm = new int[tInfo->totalTypes];
    kernelInfo->nBlocksProc = new int[tInfo->totalTypes];
}

/***************************************************************************
 * Controls the simulation and kernel calls
 ***************************************************************************/
int GpuSimulationController::launchGpuExecution() {

	defineThreadTypes();

	printf("process = %d | threadNumber = %d | types [%d|%d] | seed=%d \n", tInfo->currProcess, tInfo->threadNumber, tInfo->startTypeThread, tInfo->endTypeThread, tInfo->sharedData->globalSeed);

	/**------------------------------------------------------------------------------------
	 * Creates the neurons that will be simulated by the threads
	 *-------------------------------------------------------------------------------------*/
    createNeurons();

    int *nNeurons = tInfo->nNeurons;
    int startTypeThread = tInfo->startTypeThread;
    int endTypeThread = tInfo->endTypeThread;
    int threadNumber = tInfo->threadNumber;

    pthread_mutex_lock(sharedData->mutex);
    if(sharedData->hList == 0)
    	updateSharedDataInfo();
    pthread_mutex_unlock(sharedData->mutex);

    //Synchronize threads before starting
    syncCpuThreads();

    bench.matrixSetup = gettimeInMilli();
    bench.matrixSetupF = (bench.matrixSetup - bench.start) / 1000.;

    /*--------------------------------------------------------------
	 * Configure the Device and GPU kernel information
	 *--------------------------------------------------------------*/
    configureGpuKernel();

    /*--------------------------------------------------------------
	 * Initializes the benchmark counters
	 *--------------------------------------------------------------*/
    if(threadNumber == 0){
        bench.totalHinesKernel = 0;
        bench.totalConnRead = 0;
        bench.totalConnWait = 0;
        bench.totalConnWrite = 0;
    }
    /*--------------------------------------------------------------
	 * Allocates the memory on the GPU for neuron information and transfers the data
	 *--------------------------------------------------------------*/
    for(int type = startTypeThread;type < endTypeThread;type++){
        printf("GPU allocation with %d neurons, %d comparts on device %d thread %d process %d.\n", nNeurons[type], sharedData->matrixList[type][0].nComp, tInfo->deviceNumber, threadNumber, tInfo->currProcess);
        prepareExecution(type);
    }

    /*--------------------------------------------------------------
	 * Allocates the memory on the GPU for the communications and transfers the data
	 *--------------------------------------------------------------*/
    prepareSynapses();
    SynapticData *synData = sharedData->synData;
    int nKernelSteps = kernelInfo->nKernelSteps;

    /*--------------------------------------------------------------
	 * Sends the complete data to the GPUs
	 *--------------------------------------------------------------*/
    for(int type = startTypeThread;type < endTypeThread;type++){
        hipMalloc((void**)((((&(sharedData->hGpu[type]))))), sizeof (HinesStruct) * nNeurons[type]);
        hipMemcpy(sharedData->hGpu[type], sharedData->hList[type], sizeof (HinesStruct) * nNeurons[type], hipMemcpyHostToDevice);
        checkCUDAError("Memory Allocation:");
    }

    /*--------------------------------------------------------------
	 * Prepare the spike list in the format used in the GPU
	 *--------------------------------------------------------------*/
    int maxSpikesNeuron = 5000; //5000;
    for(int type = startTypeThread;type < endTypeThread;type++){
        int neuronSpikeListSize = maxSpikesNeuron * nNeurons[type];
        synData->spikeListGlobal[type] = (ftype*)((((malloc(sizeof (ftype) * neuronSpikeListSize)))));
        synData->weightListGlobal[type] = (ftype*)((((malloc(sizeof (ftype) * neuronSpikeListSize)))));
        hipMalloc((void**)((((&(synData->spikeListDevice[type]))))), sizeof (ftype) * neuronSpikeListSize);
        hipMalloc((void**)((((&(synData->weightListDevice[type]))))), sizeof (ftype) * neuronSpikeListSize);
        if(type == 0)
            printf("Spike List size of %.3f MB for each type.\n", sizeof (ftype) * neuronSpikeListSize / 1024. / 1024.);

    }

    /*--------------------------------------------------------------
	 * Creates the connections between the neurons
	 *--------------------------------------------------------------*/
    if (threadNumber == 0) {
		sharedData->connection = new Connections();
		sharedData->connection->connectRandom (sharedData->pyrConnRatio, sharedData->inhConnRatio,
				sharedData->typeList, tInfo->startTypeProcess, tInfo->endTypeProcess, tInfo->totalTypes,
				nNeurons, sharedData, threadNumber);

		if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == GPU_COMM) {
			sharedData->connGpuListHost   = (ConnGpu **)malloc(tInfo->totalTypes * sizeof(ConnGpu *));
			sharedData->connGpuListDevice = (ConnGpu **)malloc(tInfo->totalTypes * sizeof(ConnGpu *));
		}

		sharedData->connInfo = sharedData->connection->getMPIConnections();
	}

    /*--------------------------------------------------------------
	 * [MPI] Send the connection list to other MPI processes
	 *--------------------------------------------------------------*/
    if(threadNumber == 0)
        mpiAllGatherConnections();

    /*--------------------------------------------------------------
	 * Guarantees that all connections have been setup
	 *--------------------------------------------------------------*/
    syncCpuThreads();

    /*--------------------------------------------------------------
	 * Creates the connection list for usage in the GPU communication
	 *--------------------------------------------------------------*/
    if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == GPU_COMM)
    	createGpuCommunicationStructures();

    /*--------------------------------------------------------------
	 * Prepare the lists of generated spikes used for GPU spike delivery
	 *--------------------------------------------------------------*/
    if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == GPU_COMM)
    	prepareGpuSpikeDeliveryStructures();

    /*--------------------------------------------------------------
	 * [MPI] Prepare the genSpikeListTime to receive the values from other processes
	 *--------------------------------------------------------------*/
    if (threadNumber == 0) {
		for (int type = 0; type < tInfo->totalTypes; type++) {
			if ( type < tInfo->startTypeProcess || tInfo->endTypeProcess <= type ) {
				int spikeTimeListSize = GENSPIKETIMELIST_SIZE;
				synData->genSpikeTimeListHost[type] = (ftype *) malloc(sizeof(ftype) * spikeTimeListSize * nNeurons[type]);
				synData->nGeneratedSpikesHost[type] = (ucomp *) malloc(sizeof(ucomp) * nNeurons[type]);
			}
		}
	}
    /*--------------------------------------------------------------
	 * Synchronize threads before beginning [Used only for Benchmarking]
	 *--------------------------------------------------------------*/
    //syncCpuThreads(sharedData);
    printf("Launching GPU kernel with %d blocks and %d (+1) threads per block for types %d-%d for thread %d "
    		"on device %d [%s|%d.%d|MP=%d|G=%dMB|S=%dkB].\n", kernelInfo->nBlocksProc[startTypeThread],
    		nNeurons[startTypeThread] / kernelInfo->nBlocksProc[startTypeThread], startTypeThread, endTypeThread - 1,
    		threadNumber, tInfo->deviceNumber, tInfo->prop->name, tInfo->prop->major, tInfo->prop->minor,
    		tInfo->prop->multiProcessorCount, (int)((tInfo->prop->totalGlobalMem / 1024 / 1024)),
    		(int)((tInfo->prop->sharedMemPerBlock / 1024)));

    if(threadNumber == 0){
        bench.execPrepare = gettimeInMilli();
        bench.execPrepareF = (bench.execPrepare - bench.matrixSetup) / 1000.;
    }

    /*--------------------------------------------------------------
	 * Solves the matrix for n steps
	 *--------------------------------------------------------------*/
    ftype dt = sharedData->dt;
    int nSteps = sharedData->totalTime / dt;

    for (kStep = 0; kStep < nSteps; kStep += nKernelSteps) {

		// Synchronizes the thread to wait for the communication

		if (threadNumber == 0 && kStep % 100 == 0)
			printf("Starting Kernel %d -----------> %d \n", threadNumber, kStep);

		if (threadNumber == 0) // Benchmarking
			bench.kernelStart  = gettimeInMilli();

		for (int type = startTypeThread; type < endTypeThread; type++) {

			int nThreadsProc =  nNeurons[type]/kernelInfo->nBlocksProc[type];
			if (nNeurons[type] % kernelInfo->nBlocksProc[type]) nThreadsProc++;

			checkCUDAError("Before SolveMatrixG Kernel:");
			solveMatrixG<<<kernelInfo->nBlocksProc[type], nThreadsProc, kernelInfo->sharedMemSizeProc>>>(
					sharedData->hGpu[type], nKernelSteps, nNeurons[type],
					synData->spikeListDevice[type], synData->weightListDevice[type],
					synData->spikeListPosDevice[type], synData->spikeListSizeDevice[type],
					synData->vmListDevice[type]);
		}

		hipDeviceSynchronize();

		if (threadNumber == 0) // Benchmarking
			bench.kernelFinish = gettimeInMilli();

		/*--------------------------------------------------------------
		 * Reads information from spike sources fromGPU
		 *--------------------------------------------------------------*/
		readGeneratedSpikesFromGPU();

		/*--------------------------------------------------------------
		 * Adds the generated spikes to the target synaptic channel
		 * Used only for communication processing in the CPU
		 *--------------------------------------------------------------*/
		if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == CPU_COMM)
			addReceivedSpikesToTargetChannelCPU();

		if (threadNumber == 0 && benchConf.gpuCommMode == CPU_COMM)
			bench.connRead = gettimeInMilli();
		else if (threadNumber == 0) {
			bench.connRead = 0;
			bench.connWrite = 0;
		}
		/*--------------------------------------------------------------
		 * Synchronize threads before communication
		 *--------------------------------------------------------------*/
		syncCpuThreads();

		/*--------------------------------------------------------------
		 * [MPI] Send the list of generated spikes to other processes
		 *--------------------------------------------------------------*/
#ifdef MPI_GPU_NN
		broadcastGeneratedSpikesMPISync();
#endif

		if (threadNumber == 0)
			bench.connWait = gettimeInMilli();

		/*--------------------------------------------------------------
		 * Used to print spike statistics in the end of the simulation
		 *--------------------------------------------------------------*/
		if (threadNumber == 0)
			for (int type=0; type < tInfo->totalTypes; type++)
				for (int c=0; c<nNeurons[type]; c++)
					sharedData->spkStat->addGeneratedSpikes(type, c, NULL, synData->nGeneratedSpikesHost[type][c]);

		/*--------------------------------------------------------------
		 * Copy the Vm from GPUs to the CPU memory
		 *--------------------------------------------------------------*/
		if (benchConf.assertResultsAll == 1 || benchConf.printAllVmKernelFinish == 1)
			for (int type = startTypeThread; type < endTypeThread; type++)
				hipMemcpy(synData->vmListHost[type], synData->vmListDevice[type], sizeof(ftype) * nNeurons[type], hipMemcpyDeviceToHost);

		/*--------------------------------------------------------------
		 * Writes Vm to file at the end of each kernel execution
		 *--------------------------------------------------------------*/
		if (benchConf.assertResultsAll == 1)
			checkVmValues();

		/*--------------------------------------------------------------
		 * Check if Vm is ok for all neurons
		 *--------------------------------------------------------------*/
		if (threadNumber == 0 && benchConf.printAllVmKernelFinish == 1)
			sharedData->neuronInfoWriter->writeVmToFile(kStep);


		/*--------------------------------------------------------------
		 * Copy the generatedSpikeList to the GPUs
		 *--------------------------------------------------------------*/
		if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == GPU_COMM)
			copyGeneratedSpikeListsToGPU();

		/*-------------------------------------------------------
		 * Perform Communications
		 *-------------------------------------------------------*/
		for (int type = startTypeThread; type < endTypeThread; type++) {

			/*-------------------------------------------------------
			 *  Generates random spikes for the network
			 *-------------------------------------------------------*/
			RandomSpikeInfo randomSpkInfo;
			randomSpkInfo.listSize = sharedData->inputSpikeRate * nKernelSteps * dt * nNeurons[type] * 3;
			randomSpkInfo.spikeTimes = new ftype[ randomSpkInfo.listSize ];
			randomSpkInfo.spikeDest = new int[ randomSpkInfo.listSize ];
			generateRandomSpikes(type, randomSpkInfo);

			/*-------------------------------------------------------
			 * Perform GPU Communications
			 *-------------------------------------------------------*/
			if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == GPU_COMM)
				performGPUCommunications(type, randomSpkInfo);

			delete []randomSpkInfo.spikeTimes;
			delete []randomSpkInfo.spikeDest;

			/*-------------------------------------------------------
			 * Perform CPU Communications
			 *-------------------------------------------------------*/
			if (benchConf.gpuCommMode == CHK_COMM || benchConf.gpuCommMode == CPU_COMM)
				performCPUCommunication(type, maxSpikesNeuron, randomSpkInfo.nRandom);
		}

		if (threadNumber == 0)
			if (benchConf.gpuCommBenchMode == GPU_COMM_SIMPLE || benchConf.gpuCommMode == CPU_COMM)
				bench.connWrite = gettimeInMilli();

		if (threadNumber == 0 && benchConf.printSampleVms == 1)
			sharedData->neuronInfoWriter->writeSampleVm(kStep);

		if (benchConf.printAllSpikeTimes == 1)
			if (threadNumber == 0) // Uses only data from SpikeStatistics::addGeneratedSpikes
				sharedData->spkStat->printKernelSpikeStatistics((kStep+nKernelSteps)*dt);

		if (threadNumber == 0)
			updateBenchmark();


	}
    // --------------- Finished the simulation ------------------------------------

	if (threadNumber == 0) {
		bench.execExecution  = gettimeInMilli();
		bench.execExecutionF = (bench.execExecution - bench.execPrepare)/1000.;
	}

	if (threadNumber == 0) {
		//printf("%10.2f\t%10.5f\t%10.5f\n", dt * nSteps, (vmTimeSerie[0])[nCompVmTimeSerie*nKernelSteps-1], (vmTimeSerie[0])[nKernelSteps-1]);
		//printf("%10.2f\t%10.5f\t%10.5f\n", dt * nSteps, (vmTimeSerie[1])[nCompVmTimeSerie*nKernelSteps-1], (vmTimeSerie[1])[nKernelSteps-1]);
	}

	// Used to print spike statistics in the end of the simulation
	if (threadNumber == 0)
		sharedData->spkStat->printSpikeStatistics("spikeGpu.dat", sharedData->totalTime, bench, tInfo->startTypeProcess, tInfo->endTypeProcess);

	// TODO: Free CUDA Memory
	for (int type = startTypeThread; type < endTypeThread; type++) {
		hipFree(synData->spikeListDevice[type]);
		hipFree(synData->weightListDevice[type]);
		free(synData->spikeListGlobal[type]);
		free(synData->weightListGlobal[type]);
	}

	if (threadNumber == 0) {
		delete[] kernelInfo->nBlocksComm;
		delete[] kernelInfo->nThreadsComm;
		delete sharedData->neuronInfoWriter;
	}

	printf("Finished GPU execution.\n" );

	return 0;
}
