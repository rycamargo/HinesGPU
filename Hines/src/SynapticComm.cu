#include "hip/hip_runtime.h"
/*
 * SynapticComm.cu
 *
 *  Created on: 10/12/2010
 *      Author: rcamargo
 */

#include "Connections.hpp"
#include <cstdio>
#include <cassert>

#include <hip/hip_runtime.h> // Necessary to allow better eclipse integration
#include <hip/hip_runtime_api.h> // Necessary to allow better eclipse integration
#include <> // Necessary to allow better eclipse integration
#include <hip/device_functions.h> // Necessary to allow better eclipse integration

// Put here so that eclipse will not see this function as an error in the code
extern void __syncthreads(void);

extern void checkCUDAError(const char *msg);

// TODO: remove connections from Connections NEW
ConnGpu* createGpuConnections( ConnectionInfo *connInfo, int destType, int *nNeurons, int nGroups ) {

	// Contains the structures with the connections for each neuron group
	ConnGpu *connGpuTypeHost = (ConnGpu *)malloc(nGroups*sizeof(ConnGpu));

	int nConnectionsTotal[nGroups];
	int nNeuronsPerGroup =  nNeurons[destType]/nGroups;
	int nGroupsExtraNeuron = (nNeurons[destType] % nGroups);
	int nNeuronsExtraGroups = nGroupsExtraNeuron * (nNeuronsPerGroup + 1);

	int nNeuronsInPreviousGroups = 0;
	for (int group=0; group<nGroups; group++) {

		ConnGpu & connGpu = connGpuTypeHost[group];
		nConnectionsTotal[group] = 0;

		connGpu.nNeuronsGroup = nNeuronsPerGroup;
		connGpu.nNeuronsInPreviousGroups = nNeuronsInPreviousGroups;

		if ( group < nGroupsExtraNeuron ) connGpu.nNeuronsGroup++;

		nNeuronsInPreviousGroups += connGpu.nNeuronsGroup;
	}

	/**
	 * Counts the total number of connections for the group
	 */
	for (int conn=0; conn < connInfo->nConnections; conn++) {

		if (connInfo->dest[conn] / CONN_NEURON_TYPE == destType) {

			int destNeuron = connInfo->dest[conn] % CONN_NEURON_TYPE;
			int group = destNeuron / nNeuronsPerGroup;
			if (nNeurons[destType] % nGroups != 0) {
				if (destNeuron < nNeuronsExtraGroups)
					group = destNeuron / (nNeuronsPerGroup+1);
				else
					group = nGroupsExtraNeuron + ((destNeuron - nNeuronsExtraGroups) / nNeuronsPerGroup);
			}
			nConnectionsTotal[group]++;
		}
	}


	ftype totalConnections = 0;
	for (int group=0; group<nGroups; group++)
		totalConnections +=nConnectionsTotal[group];
	printf ("Allocated %.3f MB for %.6fM connections info for destType %d with %d groups. \n",
			totalConnections/1000/1000 * ( 2*sizeof(int) + sizeof(ucomp) + 2*sizeof(ftype) ),
			totalConnections/1000/1000, destType, nGroups);


	for (int group=0; group<nGroups; group++) {

		ConnGpu & connGpu = connGpuTypeHost[group];
		connGpu.nConnectionsTotal = nConnectionsTotal[group];

		checkCUDAError("Allocation error 0 at [SynapticComm.cfor (int neuron = 0; neuron < tInfo->nNeurons[type]; neuron++) {u]:");
		/**
		 * Allocates the memory to keep the connection information in the GPU and CPU
		 */
		hipMalloc( (void **) &(connGpu.srcDevice), 	nConnectionsTotal[group]*sizeof(int) );
		connGpu.srcHost = (int *)malloc( nConnectionsTotal[group]*sizeof(int) );
		checkCUDAError("Allocation error 1 at [SynapticComm.cu]:");

		hipMalloc( (void **) &(connGpu.destDevice), 	nConnectionsTotal[group]*sizeof(int) );
		connGpu.destHost = (int *)malloc( nConnectionsTotal[group]*sizeof(int) );
		checkCUDAError("Allocation error 2 at [SynapticComm.cu]:");

		hipMalloc( (void **) &(connGpu.synapseDevice),	nConnectionsTotal[group]*sizeof(ucomp) );
		connGpu.synapseHost = (ucomp *)malloc( nConnectionsTotal[group]*sizeof(ucomp) );
		checkCUDAError("Allocation error 3 at [SynapticComm.cu]:");

		hipMalloc( (void **) &(connGpu.weightDevice),	nConnectionsTotal[group]*sizeof(ftype) );
		connGpu.weightHost = (ftype *)malloc( nConnectionsTotal[group]*sizeof(ftype) );
		checkCUDAError("Allocation error 4 at [SynapticComm.cu]:");

		hipMalloc( (void **) &(connGpu.delayDevice),	nConnectionsTotal[group]*sizeof(ftype) );
		connGpu.delayHost = (ftype *)malloc( nConnectionsTotal[group]*sizeof(ftype) );
		checkCUDAError("Allocation error 5 at [SynapticComm.cu]:");
	}

	/**
	 * Copies the connection info data to the host memory
	 */
	int memPosList[nGroups];
	for (int group=0; group<nGroups; group++)
		memPosList[group] = 0;


	for (int conn=0; conn < connInfo->nConnections; conn++) {

		if (connInfo->dest[conn] / CONN_NEURON_TYPE == destType) {

			int destNeuron = connInfo->dest[conn] % CONN_NEURON_TYPE;
			int group = destNeuron / nNeuronsPerGroup;
			if (nNeurons[destType] % nGroups != 0) {
				if (destNeuron < nNeuronsExtraGroups)
					group = destNeuron / (nNeuronsPerGroup+1);
				else
					group = nGroupsExtraNeuron + ((destNeuron - nNeuronsExtraGroups) / nNeuronsPerGroup);
			}

			ConnGpu & connGpu = connGpuTypeHost[group];
			int memPos = memPosList[group];

			connGpu.srcHost    [memPos] = connInfo->source [conn];
			connGpu.destHost   [memPos]	= connInfo->dest   [conn];  // TODO: can move to another vector
			connGpu.synapseHost[memPos]	= connInfo->synapse[conn];  // TODO: can move to another vector
			connGpu.weightHost [memPos] = connInfo->weigth [conn];  // TODO: can move to another vector
			connGpu.delayHost  [memPos]	= connInfo->delay  [conn];  // TODO: can move to another vector

			memPosList[group]++;
		}

	}

	/**
	 * Copies the connection info data to the device memory
	 */
	for (int group=0; group<nGroups; group++) {

		assert (memPosList[group] == nConnectionsTotal[group]);

		ConnGpu & connGpu = connGpuTypeHost[group];
		hipMemcpy( connGpu.srcDevice, 		connGpu.srcHost, 	 nConnectionsTotal[group]*sizeof(int),	  hipMemcpyHostToDevice);
		hipMemcpy( connGpu.destDevice, 	connGpu.destHost, 	 nConnectionsTotal[group]*sizeof(int),	  hipMemcpyHostToDevice);
		hipMemcpy( connGpu.synapseDevice, 	connGpu.synapseHost, nConnectionsTotal[group]*sizeof(ucomp), hipMemcpyHostToDevice);
		hipMemcpy( connGpu.weightDevice, 	connGpu.weightHost,  nConnectionsTotal[group]*sizeof(ftype), hipMemcpyHostToDevice);
		hipMemcpy( connGpu.delayDevice, 	connGpu.delayHost, 	 nConnectionsTotal[group]*sizeof(ftype), hipMemcpyHostToDevice);

		checkCUDAError("Memcopy error at [SynapticComm.cu]:");
	}

	int nConnectionsAllGroups = 0;
	for (int group=0; group<nGroups; group++)
		nConnectionsAllGroups += connGpuTypeHost[group].nConnectionsTotal;
	printf ("Number of connections to type %d is %d (%dk).\n", destType, nConnectionsAllGroups, nConnectionsAllGroups/1000);


	return connGpuTypeHost;
}

__device__ void updateActivationListPos (
		ftype *activationList, ucomp activationListPosSyn, int activationListSize, int cStep,
		ftype currTime, ftype dt, ucomp synapse, ftype spikeTime, ftype delay, ftype weight, int destNeuron, int nNeurons, ftype *freeMem) {


	ftype fpos = (spikeTime + delay - currTime) / dt;

	int pos  = ( activationListPosSyn + (ucomp)fpos + 1 ) % activationListSize;
	pos       += synapse * activationListSize;

	int nextPos  = ( pos + 1 ) % activationListSize;
	nextPos       += synapse * activationListSize;

	ftype diff = fpos - (int)fpos;

	//cStep = -1; // some race conditions can occur in this version and some spikes may be lost
	if (cStep < 0) {
		activationList[    pos * nNeurons + destNeuron] += (weight / dt) * ( 1 - diff );
		activationList[nextPos * nNeurons + destNeuron] += (weight / dt) * diff;

	}
	else {
		pos     =     pos * nNeurons + destNeuron;
		nextPos = nextPos * nNeurons + destNeuron;

		ftype posValue     = (weight / dt) * ( 1 - diff );
		ftype nextPosValue = (weight / dt) * diff;

		ftype *posValueO     = freeMem;
		ftype *nextPosValueO = posValueO + blockDim.x;
		int *posToUpdate     = (int *)(nextPosValueO + blockDim.x);
		int *posToUpdateO    = posToUpdate  + blockDim.x;
		int *cStepThread     = posToUpdateO + blockDim.x;


		posToUpdate[threadIdx.x]   = pos;
		posToUpdateO[threadIdx.x]  = pos;
		cStepThread[threadIdx.x]   = cStep; // to solve the problem with threads that do not enter the function
		posValueO[threadIdx.x]     = posValue;
		nextPosValueO[threadIdx.x] = nextPosValue;

		for (int i=threadIdx.x + 1; i<blockDim.x; i++)
			if ( cStep == cStepThread[i] && pos == posToUpdateO[i] ) {
				posValue     += posValueO[i];
				nextPosValue += nextPosValueO[i];
				posToUpdate[i] += 1; // just need to change the value by any amount
			}

		//__syncthreads();

		if (pos == posToUpdate[threadIdx.x]) {
			activationList[    pos] += posValue;
			activationList[nextPos] += nextPosValue;
		}
	}

}


/**
 * Updates the global activation list
 * TODO: change ConnGpu connGpuDev to reference
 */
__device__ void updateActivationList( HinesStruct *hList,
		int nNeurons, ConnGpu *connGpuListDev,
		ftype **genSpikeTimeListDev, ucomp **nGeneratedSpikesDev,
		ftype *randomSpikeTimesDev,  int *randomSpikeDestDev, int nRandom, ftype *freeMem) {

	ConnGpu connGpuDev = connGpuListDev[blockIdx.x];

	int cStep = 0;

	int spikeTimeListSize = GENSPIKETIMELIST_SIZE;

	int nNeuronsPrev  = connGpuDev.nNeuronsInPreviousGroups;
	int nSynapses = hList[0].synapseListSize; //nNeuronsPrev + threadIdx.x

	ucomp *activationListPos  = (ucomp *)freeMem; //hList[neuron].activationListPos;
	freeMem = (ftype *)(activationListPos + connGpuDev.nNeuronsGroup * nSynapses);
	if (threadIdx.x < connGpuDev.nNeuronsGroup)
		for (int i=0; i<nSynapses; i++)
			activationListPos[nSynapses*threadIdx.x + i] = hList[nNeuronsPrev + threadIdx.x].activationListPos[i];

	__syncthreads();

	ftype *activationList  = hList[0].activationList;     // global list
	int activationListSize = hList[0].activationListSize; // global value

	ftype dt = hList[0].dt;
	ftype currTime = hList[0].currStep * dt;

	for (int iConn = 0; iConn < connGpuDev.nConnectionsTotal; iConn += blockDim.x) {

		cStep = (connGpuDev.nConnectionsTotal + iConn) * GENSPIKETIMELIST_SIZE; // The value will never repeat

		if (iConn+threadIdx.x < connGpuDev.nConnectionsTotal) {

			int srcNeuron = connGpuDev.srcDevice[iConn+threadIdx.x];
			int srcType   = srcNeuron/CONN_NEURON_TYPE;
			srcNeuron = srcNeuron%CONN_NEURON_TYPE;
			int nSpikesSource = nGeneratedSpikesDev[ srcType ][ srcNeuron ];

			ftype weight   = connGpuDev.weightDevice [iConn+threadIdx.x];
			int destNeuron = connGpuDev.destDevice   [iConn+threadIdx.x]%CONN_NEURON_TYPE;
			ucomp synapse  = connGpuDev.synapseDevice[iConn+threadIdx.x];
			ftype delay    = connGpuDev.delayDevice  [iConn+threadIdx.x];

			ftype *genSpikeTimes = genSpikeTimeListDev[srcType] + spikeTimeListSize * srcNeuron;

			ucomp activationListPosSyn = activationListPos[ nSynapses * (destNeuron-nNeuronsPrev) + synapse ];

			for (int i = 0; i < nSpikesSource; i++, cStep++) {

				updateActivationListPos( activationList, activationListPosSyn, activationListSize,
						cStep, currTime, dt, synapse, genSpikeTimes[i], delay, weight, destNeuron, nNeurons, freeMem );
			}
		}
	}

	__syncthreads();

	/**
	 * Copy the random spikes
	 * Supposes that random spikes are delivered to synapse 0
	 */
	int iRnd = 0;
	cStep = 1;
	int destNeuron = -1;
	int maxNeuron  = connGpuDev.nNeuronsGroup + connGpuDev.nNeuronsInPreviousGroups;

	while (iRnd + threadIdx.x < nRandom  && destNeuron < maxNeuron) {

		destNeuron = randomSpikeDestDev[ iRnd+threadIdx.x ] % CONN_NEURON_TYPE;
		ucomp activationListPosSyn = activationListPos[ nSynapses * (destNeuron-nNeuronsPrev) + 0 ];

		if ( destNeuron >= connGpuDev.nNeuronsInPreviousGroups && destNeuron < maxNeuron ) {
			// synapse=0, delay=0, weight = 1
			updateActivationListPos( activationList, activationListPosSyn, activationListSize, cStep,
					currTime, dt, 0, randomSpikeTimesDev[iRnd+threadIdx.x], 0, 1, destNeuron, nNeurons, freeMem );
		}
		iRnd += blockDim.x;
		cStep++;
	}

}


__global__ void performCommunicationsG(int nNeurons, ConnGpu *connGpuListDev,
		ucomp **nGeneratedSpikesDev, ftype **genSpikeTimeListDev,
		HinesStruct *hList, ftype *randomSpikeTimesDev, int *randomSpikeDestDev, int nRandom) {

	extern __shared__ ftype sharedMem[];
	ftype *freeMem =sharedMem;

//	int *nReceivedSpikesShared = (int *)sharedMem;
//	int *nSpikesToKeepShared = nReceivedSpikesShared + connGpuDev.nNeuronsGroup * nSynapses;
//	int *sharedMemNext = nSpikesToKeepShared + connGpuDev.nNeuronsGroup * nSynapses;
//	int neuron = connGpuDev.nNeuronsInPreviousGroups + threadIdx.x;

	updateActivationList( hList, nNeurons, connGpuListDev, genSpikeTimeListDev, nGeneratedSpikesDev,
			randomSpikeTimesDev, randomSpikeDestDev, nRandom, freeMem);
}

